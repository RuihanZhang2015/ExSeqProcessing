#include "spdlog/spdlog.h"

#include "gpudevice.h"

namespace cudautils {

int get_gpu_num() {
    int num_gpus;

    hipGetDeviceCount(&num_gpus);
    if (num_gpus < 1) {
        auto logger = spdlog::get("mex_logger");
        logger->error("cudautils::get_gpu_num cannot get # of gpus");
        hipDeviceReset();
        return -1;
    }

    hipDeviceReset();
    return num_gpus;
}

void get_gpu_mem_size(size_t& free_size, size_t& total_size) {
    hipMemGetInfo(&free_size, &total_size);
    hipDeviceReset();
}

void resetDevice() {
    hipDeviceReset();
}

}

