#include "hip/hip_runtime.h"
/*=================================================================
 * cmd_sift_cuda.cu - perform sift on volume image data 
 *
 *  sift_cuda(vol_image, map)
 *
 *  Input:
 *    vol_image:  volume image data
 *    map:        mask map data (1: mask, 0: hole)
 *
 *  Output:
 *    ?
 *
 *=================================================================*/
 

#include <fstream>
#include <vector>
#include <iterator>
#include <algorithm>

#include "sift.h"
#include "mexutil.h"
#include "sift_bridge.h"
/*#include "sift_types.h"*/
#include "gpudevice.h"

#include "cuda_task_executor.h"

#include "spdlog/spdlog.h"
#include "stdlib.h"

int main(int argc, char* argv[]) {

    /*if (argc < 4) {*/
        /*std::cout << "Usage: " << argv[0] << " [in image file] [in mask map file] [out interpolated image file]" << std::endl;*/
        /*return 1;*/
    /*}*/

    std::shared_ptr<spdlog::logger> logger;
    try {
        spdlog::set_async_mode(4096, spdlog::async_overflow_policy::block_retry, nullptr, std::chrono::seconds(2));
        spdlog::set_level(spdlog::level::trace);
        logger = spdlog::get("mex_logger");
        if (logger == nullptr) {
            logger = spdlog::basic_logger_mt("mex_logger", "logs/mex.log");
        }
        logger->flush_on(spdlog::level::err);
        //logger->flush_on(spdlog::level::info);
    } catch (const spdlog::spdlog_ex& ex) {
        std::cout << "Log initialization failed: " << ex.what() << std::endl;
        return 1;
    }

    try {
        logger->info("{:=>50}", " sift_cuda start");

        /*std::string in_image_filename1(argv[1]);*/
        /*std::string in_map_filename2  (argv[2]);*/
        /*std::string in_image_filename1("img_2kypts.bin");*/
        std::string in_image_filename1("image_ones.bin");
        std::string in_map_filename2  ("map_2kypts.bin");
        unsigned int x_size, y_size, z_size, x_size1, y_size1, z_size1;
        /*x_size = atoi(argv[4]);*/
        /*y_size = atoi(argv[5]);*/
        /*z_size = atoi(argv[6]);*/

        int keypoint_num;
        try {
            keypoint_num = atoi(argv[1]);
        } catch (const spdlog::spdlog_ex& ex) {
            std::cout << "Must provide # of keypoints `$./sift_cuda 1` " << ex.what() << std::endl;
            return 1;
        }

        logger->info("# of keypoints = {}", keypoint_num);
        x_size = 2048;
        y_size = 2048;
        z_size = 251;
        x_size1 = x_size;
        y_size1 = y_size;
        z_size1 = z_size;

        /*unsigned int x_size, y_size, z_size, x_size1, y_size1, z_size1;*/
        std::ifstream fin1(in_image_filename1, std::ios::binary);
        /*fin1.read((char*)&x_size, sizeof(unsigned int));*/
        /*fin1.read((char*)&y_size, sizeof(unsigned int));*/
        /*fin1.read((char*)&z_size, sizeof(unsigned int));*/

        std::ifstream fin2(in_map_filename2, std::ios::binary);
        /*fin2.read((char*)&x_size1, sizeof(unsigned int));*/
        /*fin2.read((char*)&y_size1, sizeof(unsigned int));*/
        /*fin2.read((char*)&z_size1, sizeof(unsigned int));*/

        if (x_size != x_size1 || y_size != y_size1 || z_size != z_size1) {
            logger->error("the dimension of image and map is not the same. image({},{},{}), map({},{},{})",
                    x_size, y_size, z_size, x_size1, y_size1, z_size1);
            fin1.close();
            fin2.close();
            return 1;
        }

        // create image
        long image_size = x_size * y_size * z_size;
        double* in_image = (double*) malloc(image_size * sizeof(double));
        int8_t* in_map = (int8_t*) malloc(image_size * sizeof(int8_t));
        for (long i=0; i < image_size; i++) {
            in_image[i] = rand() % 100 + 1.0;
            in_map[i] = 1.0;
        }

        // create map
        long long idx;
        for (int i=0; i < keypoint_num; i++) {
            // warning not evenly distributed across the image
            idx = (x_size * rand()) % image_size;
            in_map[idx] = 0.0; // select this point for processing
        }

        /*fin2.read((char*)in_map  .data(), image_size * sizeof(int8_t));*/
        fin1.close();
        fin2.close();

        const unsigned int num_streams = 20;
        int num_gpus = cudautils::get_gpu_num();
        logger->info("# of gpus = {}", num_gpus);
        logger->info("# of streams = {}", num_streams);
        logger->info("# of keypoints = {}", keypoint_num);

        /*std::vector<double> out_interp_image(x_size * y_size * z_size);*/

        const unsigned int x_sub_size = min(2048, x_size);
        const unsigned int y_sub_size = min(2048, y_size / num_gpus);
        const unsigned int dx = min(256, x_sub_size);
        const unsigned int dy = min(256, y_sub_size);
        const unsigned int dw = 0;

        cudautils::SiftParams sift_params;
        double* fv_centers = sift_defaults(&sift_params,
                x_size, y_size, z_size, keypoint_num);

        logger->info("x_size={},y_size={},z_size={},x_sub_size={},y_sub_size={},dx={},dy={},dw={}",
                x_size, y_size, z_size, x_sub_size, y_sub_size, dx, dy, dw);

        try {
            cudautils::Keypoint_store keystore;

            cudautils::sift_bridge(
                    logger, x_size, y_size, z_size, x_sub_size, y_sub_size, dx,
                    dy, dw, num_gpus, num_streams, in_image, in_map,
                    sift_params, fv_centers, &keystore);

            /*std::shared_ptr<cudautils::Sift> ni =*/
                /*std::make_shared<cudautils::Sift>(x_size, y_size, z_size,*/
                        /*x_sub_size, y_sub_size, dx, dy, dw, num_gpus,*/
                        /*num_streams, sift_params, fv_centers);*/

            /*cudautils::CudaTaskExecutor executor(num_gpus, num_streams, ni);*/

            /*logger->info("setImage start");*/
            /*ni->setImage(in_image);*/
            /*logger->info("setImage end");*/
            /*ni->setMapToBeInterpolated(in_map);*/
            /*logger->info("setMap end");*/

            /*logger->info("calc start");*/
            /*executor.run();*/
            /*logger->info("calc end");*/

            /*logger->info("getKeystore start");*/
            /*ni->getKeystore(&keystore);*/
            /*logger->info("getKeystore end");*/

            /*mxArray* mxKeystore;*/
            /*// Convert the output keypoints*/
            /*if ((mxKeystore = kp2mx(&keystore, sift_params)) == NULL)*/
                /*logger->error("keystore to mex error occurred");*/

            free(in_image);
            free(in_map);

        } catch (...) {
            logger->error("internal unknown error occurred");
        }

        logger->info("{:=>50}", " sift_cuda end");

        logger->flush();
        spdlog::drop_all();
    } catch (...) {
        logger->flush();
        throw;
    }

    return 0;
}

