#include "hip/hip_runtime.h"
/*=================================================================
 * cmd_sift_cuda.cu - perform sift on volume image data 
 *
 *  sift_cuda(vol_image, map)
 *
 *  Input:
 *    vol_image:  volume image data
 *    map:        mask map data (1: mask, 0: hole)
 *
 *  Output:
 *    ?
 *
 *=================================================================*/
 

#include <fstream>
#include <vector>

#include "sift.h"
#include "gpudevice.h"

#include "cuda_task_executor.h"

#include "spdlog/spdlog.h"


int main(int argc, char* argv[]) {

    if (argc < 4) {
        std::cout << "Usage: " << argv[0] << " [in image file] [in mask map file] [out interpolated image file]" << std::endl;
        return 1;
    }

    std::shared_ptr<spdlog::logger> logger;
    try {
        spdlog::set_async_mode(4096, spdlog::async_overflow_policy::block_retry, nullptr, std::chrono::seconds(2));
        spdlog::set_level(spdlog::level::trace);
        logger = spdlog::get("mex_logger");
        if (logger == nullptr) {
            logger = spdlog::basic_logger_mt("mex_logger", "logs/mex.log");
        }
        logger->flush_on(spdlog::level::err);
        //logger->flush_on(spdlog::level::info);
    } catch (const spdlog::spdlog_ex& ex) {
        std::cout << "Log initialization failed: " << ex.what() << std::endl;
        return 1;
    }

    try {
        logger->info("{:=>50}", " sift_cuda start");

        std::string in_image_filename1(argv[1]);
        std::string in_map_filename2  (argv[2]);
        std::string out_interp_image_filename(argv[3]);

        unsigned int x_size, y_size, z_size, x_size1, y_size1, z_size1;
        std::ifstream fin1(in_image_filename1, std::ios::binary);
        fin1.read((char*)&x_size, sizeof(unsigned int));
        fin1.read((char*)&y_size, sizeof(unsigned int));
        fin1.read((char*)&z_size, sizeof(unsigned int));

        std::ifstream fin2(in_map_filename2, std::ios::binary);
        fin2.read((char*)&x_size1, sizeof(unsigned int));
        fin2.read((char*)&y_size1, sizeof(unsigned int));
        fin2.read((char*)&z_size1, sizeof(unsigned int));

        if (x_size != x_size1 || y_size != y_size1 || z_size != z_size1) {
            logger->error("the dimension of image and map is not the same. image({},{},{}), map({},{},{})",
                    x_size, y_size, z_size, x_size1, y_size1, z_size1);
            fin1.close();
            fin2.close();
            return 1;
        }

        std::vector<double> in_image(x_size * y_size * z_size);
        std::vector<int8_t> in_map  (x_size * y_size * z_size);
        fin1.read((char*)in_image.data(), x_size * y_size * z_size * sizeof(double));
        fin2.read((char*)in_map  .data(), x_size * y_size * z_size * sizeof(int8_t));
        fin1.close();
        fin2.close();

//        int num_gpus = 1;
        int num_gpus = cudautils::get_gpu_num();
        logger->info("# of gpus = {}", num_gpus);

        std::vector<double> out_interp_image(x_size * y_size * z_size);

        const unsigned int x_sub_size = min(2048, x_size);
        const unsigned int y_sub_size = min(1024, y_size);
        const unsigned int dx = min(256, x_sub_size);
        const unsigned int dy = min(256, y_sub_size);
        const unsigned int dw = 2;

        const unsigned int num_streams = 20;
        logger->info("x_size={},y_size={},z_size={},x_sub_size={},y_sub_size={},dx={},dy={},dw={},# of streams={}",
                x_size, y_size, z_size, x_sub_size, y_sub_size, dx, dy, dw, num_streams);

        try {
            std::shared_ptr<cudautils::Sift> ni =
                std::make_shared<cudautils::Sift>(x_size, y_size, z_size, x_sub_size, y_sub_size, dx, dy, dw, num_gpus, num_streams);

            cudautils::CudaTaskExecutor executor(num_gpus, num_streams, ni);

            logger->info("setImage start");
            ni->setImage(in_image);
            logger->info("setImage end");
            ni->setMapToBeInterpolated(in_map);
            logger->info("setMap end");

            logger->info("calc start");
            executor.run();
            logger->info("calc end");

            logger->info("getImage start");
            ni->getImage(out_interp_image);
            logger->info("getImage end");

            logger->info("saveImage start");
            std::ofstream fout(out_interp_image_filename, std::ios::binary);
            fout.write((char*)&x_size, sizeof(unsigned int));
            fout.write((char*)&y_size, sizeof(unsigned int));
            fout.write((char*)&z_size, sizeof(unsigned int));

            fout.write((char*)out_interp_image.data(), x_size * y_size * z_size * sizeof(double));
            fout.close();
            logger->info("saveImage end");

        } catch (...) {
            logger->error("internal unknown error occurred");
        }

        logger->info("{:=>50}", " sift_cuda end");

        logger->flush();
        spdlog::drop_all();
    } catch (...) {
        logger->flush();
        throw;
    }

    return 0;
}

