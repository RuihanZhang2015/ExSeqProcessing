#include "hip/hip_runtime.h"
#include <iostream>
#include <future>

#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>

#include <hip/hip_runtime.h>
#include <cmath>
#include <numeric> //std::inner_product

#include "sift.h"
#include "matrix_helper.h"
#include "cuda_timer.h"

#include "spdlog/spdlog.h"


namespace cudautils {

struct Keypoint {
    int x;
    int y;
    int z;
    double xyScale;
    double tScale;
    int* ivec; //stores the flattened descriptor vector
};

struct SiftParams {
    double MagFactor;
    int IndexSize;
    int nFaces;
    int Tessellation_levels;
    int Smooth_Flag;
    double SigmaScaled;
    double Tessel_thresh;
    double Smooth_Var;
    int TwoPeak_Flag;
    double xyScale;
    double tScale;
    double MaxIndexVal;
    //FIXME must be in row order
    double* fv_centers;
    int fv_centers_len;
};

/*struct FV {*/
    /*double* vertices;*/
    /*double* faces;*/
    /*double* centers;*/
/*};*/

/*FV sphere_tri(int maxlevel, int r) {*/
    /*
     sphere_tri - generate a triangle mesh approximating a sphere
    
     Usage: FV = sphere_tri(Nrecurse,r)
    
       Nrecurse is int >= 0, setting the recursions (default 0)
    
       r is the radius of the sphere (default 1)
    
       FV has fields FV.vertices and FV.faces.  The vertices
       are listed in clockwise order in FV.faces, as viewed
       from the outside in a RHS coordinate system.
    
     The function uses recursive subdivision.  The first
     approximation is an icosahedron. Each level of refinement
     subdivides each triangle face by a factor of 4 (see also
     mesh_refine). At each refinement, the vertices are
     projected to the sphere surface (see sphere_project).
    
     A recursion level of 3 or 4 is a good sphere surface, if
     gouraud shading is used for rendering.
    
     The returned struct can be used in the patch command, eg:
    
      create and plot, vertices: [2562x3] and faces: [5120x3]
     FV = sphere_tri('ico',4,1);
     lighting phong; shading interp; figure;
     patch('vertices',FV.vertices,'faces',FV.faces,...
           'facecolor',[1 0 0],'edgecolor',[.2 .2 .6]);
     axis off; camlight infinite; camproj('perspective');
    
     See also: mesh_refine, sphere_project
    
     Cuda revision of:
     Licence:  GNU GPL, no implied or express warranties
     Jon Leech (leech @ cs.unc.edu) 3/24/89
     icosahedral code added by Jim Buddenhagen (jb1556@daditz.sbc.com) 5/93
     06/2002, adapted from c to matlab by Darren.Weber_at_radiology.ucsf.edu
     05/2004, reorder of the faces for the 'ico' surface so they are indeed
     clockwise!  Now the surface normals are directed outward.  Also reset the
     default recursions to zero, so we can get out just the platonic solids.
    
    */

    /*// default maximum subdivision level*/
    /*if (maxlevel < 0)*/
        /*maxlevel = 0;*/

    /*// default radius*/
    /*if (r < 0)*/
        /*r = 1;*/

    /*// define the icosehedron*/

    /*// Twelve vertices of icosahedron on unit sphere*/
    /*double tau = 0.8506508084; // t=(1+sqrt(5))/2, tau=t/sqrt(1+t^2)*/
    /*double one = 0.5257311121; // one=1/sqrt(1+t^2) , unit sphere*/

    /*FV fv;*/
    
    /*// store the vertices in column (Matlab) order */
    /*fv.vertices = new double[12][3];*/
    /*fv.vertices[ 0] = {  tau,  one,    0 }; // ZA*/
    /*fv.vertices[ 1] = { -tau,  one,    0 }; // ZB*/
    /*fv.vertices[ 2] = { -tau, -one,    0 }; // ZC*/
    /*fv.vertices[ 3] = {  tau, -one,    0 }; // ZD*/
    /*fv.vertices[ 4] = {  one,   0 ,  tau }; // YA*/
    /*fv.vertices[ 5] = {  one,   0 , -tau }; // YB*/
    /*fv.vertices[ 6] = { -one,   0 , -tau }; // YC*/
    /*fv.vertices[ 7] = { -one,   0 ,  tau }; // YD*/
    /*fv.vertices[ 8] = {   0 ,  tau,  one }; // XA*/
    /*fv.vertices[ 9] = {   0,  -tau,  one }; // XB*/
    /*fv.vertices[10] = {   0 , -tau, -one }; // XC*/
    /*fv.vertices[11] = {   0 ,  tau, -one }; // XD*/
    
    /*// Structure for unit icosahedron*/
    /*// Fixme check this is in correct col order*/
    /*// previous matlab code was in ; order*/
    /*fv.faces = {  5,  8,  9 ,*/
               /*5, 10,  8 ,*/
               /*6, 12,  7 ,*/
               /*6,  7, 11 ,*/
               /*1,  4,  5 ,*/
               /*1,  6,  4 ,*/
               /*3,  2,  8 ,*/
               /*3,  7,  2 ,*/
               /*9, 12,  1 ,*/
               /*9,  2, 12 ,*/
              /*10,  4, 11 ,*/
              /*10, 11,  3 ,*/
               /*9,  1,  5 ,*/
              /*12,  6,  1 ,*/
               /*5,  4, 10 ,*/
               /*6, 11,  4 ,*/
               /*8,  2,  9 ,*/
               /*7, 12,  2 ,*/
               /*8, 10,  3 ,*/
               /*7,  3, 11 };*/
    


    /*// -----------------*/
    /*// refine the starting shapes with subdivisions*/
    /*if maxlevel,*/
        
        /*// Subdivide each starting triangle (maxlevel) times*/
        /*for level = 1:maxlevel,*/
            
            /*// Subdivide each triangle and normalize the new points thus*/
            /*// generated to lie on the surface of a sphere radius r.*/
            /*fv = mesh_refine_tri4(fv);*/
            /*fv.vertices = sphere_project(fv.vertices,r);*/
            
            /*// An alternative might be to define a min distance*/
            /*// between vertices and recurse or use fminsearch*/
            
        /*end*/
    /*end*/

    /*for (int i=0; i < length(fv.faces); i++) {*/
        /*fv.centers(i,:) = mean(fv.vertices(fv.faces(i,:),:));*/
        /*// Unit Normalization*/
        /*fv.centers(i,:) = fv.centers(i,:) ./ sqrt(dot(fv.centers(i,:),fv.centers(i,:)));*/
    /*}*/
/*}*/

__device__
place_in_index(double* index, double mag, int i, int j, int s, 
        double* yy, double* ix, double* sift_params) {

    double tmpsum = 0.0;
    /*FIXME*/
    /*int bin_index = bin_sub2ind(i,j,s);*/
    if (sift_params.Smooth_Flag) {
        for (int tessel = 0; tessel < sift_params.Tessel_thresh; tessel++) {
            tmpsum += pow(yy[tessel], sift_params.Smooth_Var);
        }

        // Add three nearest tesselation faces
        for (int ii=0; ii<sift_params.Tessel_thresh; ii++) {
            index[bin_index] +=  mag * pow(yy[ii], sift_params.Smooth_Var ) / tmpsum;
        }
    }
        index[bin_index] += mag;
    }

}

/*r, c, s is the pixel index (x, y, z dimensions respect.) in the image within the radius of the */
/*keypoint before clamped*/
/*For each pixel, take a neighborhhod of xyradius and tiradius,*/
/*bin it down to the sift_params.IndexSize dimensions*/
/*thus, i_indx, j_indx, s_indx represent the binned index within the radius of the keypoint*/
__device__
void add_sample(double* index, double* image, double distsq, int
        r, int c, int s, int i_indx, int j_indx, int s_indx, FV
        SiftParams sift_params) {

    double sigma = sift_params.SigmaScaled;
    double weight = exp(-(distsq / (2.0 * sigma * sigma)));

    double mag;
    double* vect, yy, ix;
    /*gradient and orientation vectors calculated from 3D halo/neighboring pixels*/
    get_grad_ori_vector(image,r,c,s, mag, vect, yy, ix, sift_params);
    double mag = weight * mag; // scale magnitude by gaussian 

    place_in_index(index, mag, i_indx, j_indx, s_indx, yy, ix, sift_params);
}

// assumes r,c,s lie within accessible image boundaries
__device__
void get_grad_ori_vector(double* image, int r, int c, int s, 
        double mag, double* vect, double* yy, double* ix, SiftParams sift_params) {

    //FIXME subscripts to linear ind
    double xgrad = image[r,c+1,s] - image[r,c-1,s];
    //FIXME is this correct direction?
    double ygrad = image[r-1,c,s] - image[r+1,c,s];
    double zgrad = image[r,c,s+1] - image[r,c,s-1];

    double mag = sqrt(xgrad * xgrad + ygrad * ygrad + zgrad * zgrad);

    if (mag !=0)
        vect = {xgrad / mag, ygrad / mag, zgrad / mag};
    else
        vect = {1 0 0};
    end

    //Find the nearest tesselation face indices
    //FIXME hipblasSgemm() higher performance
    int N = sift_params.fv_centers_len;
    double corr_array[N];
    for (int i=0; i < N; i++) {
        corr_array[i] = std::inner_product(sift_params.fv_centers[i],
                sift_params.fv_centers[i] + 3,
                vect, 0.0);
    }
    int ix[N]; 
    ix = thrust::sequence(thrust::host, ix, ix + N);
    // descending order by ori_hist
    thrust::sort_by_key(thrust::host, ix, ix + N, corr_array, thrust::greater<int>());
    yy = corr_array;
}

// floor quotient, add 1
// clamp bin idx to IndexSize
inline int get_bin_idx(int orig, int radius, int IndexSize) {
    int idx = (int) 1 + ((orig + radius) / (2.0 * radius / IndexSize));
    if (idx > IndexSize)
        idx = IndexSize;
    return idx;
}

double* key_sample(key, image, sift_params) {

    /*FV fv = sphere_tri(sift_params.Tessellation_levels,1);*/

    xySpacing = key.xyScale * sift_params.MagFactor;
    tSpacing = key.tScale * sift_params.MagFactor;

    int xyiradius = round(1.414 * xySpacing * (sift_params.IndexSize + 1) / 2.0);
    int tiradius = round(1.414 * tSpacing * (sift_params.IndexSize + 1) / 2.0);

    int N = sift_params.IndexSize * sift_params.IndexSize * sift_params.IndexSize * sift_params.nFaces;
    double* index = (double*) calloc(N, sizeof(double));

    int r, c, t;
    for (int i = -xyiradius; i <= xyiradius; i++) {
        for (int j = -xyiradius; j <= xyiradius; j++) {
            for (int k = -tiradius; j <= tiradius; k++) {

                distsq = (double) i^2 + j^2 + s^2;

                // Find bin idx
                // FIXME check correct
                i_bin = get_bin_idx(i, xyiradius, sift_params.IndexSize);
                j_bin = get_bin_idx(j, xyiradius, sift_params.IndexSize);
                k_bin = get_bin_idx(k, tiradius, sift_params.IndexSize);
                
                // Find original image pixel idx
                r = key.x + i;
                c = key.y + j;
                t = key.z + k;

                // only add if within image range
                if !(r < 0  ||  r >= sift_params.image_size[0] ||
                        c < 0  ||  c >= sift_params.image_size[1]
                        || t < 0 || t >= sift_params.image_size[2]) {
                    add_sample(index, image, distsq, r, c, t,
                            i_bin, j_bin, k_bin, sift_params);
                }
            }
        }
    }

    return index;
}

double* build_ori_hists(x, y, z, radius, image, sift_params) {

    double* ori_hist = (double*) calloc(sift_params.nFaces,sizeof(double));

    double mag;
    double* vect, yy, ix;
    int r, c, t;
    for (int i = -radius; i <= radius; i++) {
        for (int j = -radius; j <= radius; j++) {
            for (int k = -radius; j <= radius; k++) {
                // Find original image pixel idx
                r = x + i;
                c = y + j;
                t = z + k;

                // only add if within image range
                if !(r < 0  ||  r >= sift_params.image_size[0] ||
                        c < 0  ||  c >= sift_params.image_size[1]
                        || t < 0 || t >= sift_params.image_size[2]) {
                    /*gradient and orientation vectors calculated from 3D halo/neighboring pixels*/
                    get_grad_ori_vector(image,r,c,t, mag, vect, yy, ix, sift_params);
                    ori_hist[ix[0]] += mag;
                }
            }
        }
    }
    return ori_hist;
}

void normalize_vec(double* vec, int len) {

    double sqlen = 0.0;
    for (int i=0; i < len; i++) {
        sqlen += vec[i] * vec[i];
    }

    double fac = 1.0 / sqrt(sqlen);
    for (int i=0; i < len; i++) {
        vec[i] = vec[i] * fac;
    }
}

Keypoint make_keypoint_sample(key, image, sift_params) {

    //FIXME add to sift_params from Matlab side
    sift_params.MaxIndexVal = 0.2;
    changed = 0;

    //FIXME make sure vec is in column order
    double* vec = key_sample(key, image, sift_params);
    VecLength = length(vec);

    vec = normalize_vec(vec, VecLength);

    for (int i=0; i < VecLength; i++) {
        if (vec[i] > sift_params.MaxIndexVal)
            vec[i] = sift_params.MaxIndexVal;
            changed = 1;
        }
    }

    if (changed) {
        vec = normalize_vec(vec, VecLength);
    }

    int intval;
    for (int i=0; i < VecLength; i++) {
        intval = round(512.0 * vec[i]);
        key.ivec[i] = (int) min(255, intval);
    }
}


Keypoint make_keypoint(double* image, int x, int y, int z, SiftParams sift_params) {
    k.x = x;
    k.y = y;
    k.z = z;
    k.xyScale = sift_params.xyScale;
    k.tScale = sift_params.tScale;
    return make_keypoint_sample(k, image, sift_params);
}

/* Main function of 3DSIFT Program from http://www.cs.ucf.edu/~pscovann/

Inputs:
image - a 3 dimensional matrix of double
xyScale and tScale - affects both the scale and the resolution, these are
usually set to 1 and scaling is done before calling this function
x, y, and z - the location of the center of the keypoint where a descriptor is requested

Outputs:
keypoint - the descriptor, varies in size depending on values in LoadParams.m
reRun - a flag (0 or 1) which is set if the data at (x,y,z) is not
descriptive enough for a good keypoint
*/
Keypoint create_descriptor(double* image, int x, int y, 
        int z, SiftParams sift_params) {

    reRun = 0;

    int radius = round(sift_params.xyScale * 3.0);

    /*FV fv = sphere_tri(sift_params.Tessellation_levels, 1);*/
    int ori_hist_len = sift_params.nFaces;
    int ix[ori_hist_len]; 
    ix = thrust::sequence(thrust::host, ix, ix + ori_hist_len);
    double* ori_hist = build_ori_hists(x, y, z, radius, image, sift_params);
    // descending order by ori_hist
    thrust::sort_by_key(thrust::host, ix, ix + ori_hist_len, ori_hist, thrust::greater<int>());
        
    if (sift_params.TwoPeak_Flag &&
            //FIXME must be in row order
            std::inner_product(sift_params.fv_centers[ix[0]],
                sift_params.fv_centers[ix[0]]
                + 3, sift_params.fv_centers[ix[1]], 0.0) > .9 &&
            std::inner_product(sift_params.fv_centers[ix[0]],
                sift_params.fv_centers[ix[0]] + 3, 
                sift_params.fv_centers[ix[2]], 0.0) > .9) {
        reRun = 1;
        return Null;
    }

    return make_keypoint(image, x, y, z, sift_params);
}

// interpolate image data
//
__global__
void interpolate_volumes(
        unsigned int x_stride,
        unsigned int y_stride,
        unsigned int map_idx_size,
        unsigned int *map_idx,
        int8_t *map,
        double *image,
        double *interpolated_values) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= map_idx_size) return;

    /*map_idx[i] linear idx of 0 value for this thread's 0 element */
    unsigned int idx_zplane = map_idx[i] - 1 - x_stride - (x_stride * y_stride); // move current pos idx by (-1, -1, -1)
    unsigned int idx = idx_zplane;

    int sum_idx = 0;
    double sum = 0.0;

    // (-1, -1, -1)  ->  (1, -1, -1)
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 0, -1)  ->  (1, 0, -1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 1, -1)  ->  (1, 1, -1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    idx_zplane += x_stride * y_stride;
    idx = idx_zplane;

    // (-1, -1, 0)  ->  (1, -1, 0)
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 0, 0)  ->  (1, 0, 0)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 1, 0)  ->  (1, 1, 0)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    idx_zplane += x_stride * y_stride;
    idx = idx_zplane;

    // (-1, -1, 1)  ->  (1, -1, 1)
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 0, 1)  ->  (1, 0, 1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 1, 1)  ->  (1, 1, 1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    if (sum_idx > 0) {
        interpolated_values[i] = sum / double(sum_idx);
    } else {
        idx_zplane = map_idx[i] - 2 * (1 + x_stride + (x_stride * y_stride)); // move current pos idx by (-2, -2, -2)

        // (u, v, w) <- (x, y, z)
        // u=0-4 v=0-4 w=0,4
        idx = idx_zplane;
        for (unsigned int v = 0; v < 5; v++) {
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
        }
        idx = idx_zplane + 4 * x_stride * y_stride;
        for (unsigned int v = 0; v < 5; v++) {
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
        }


        // u=0-4 v=0,4 w=1-3
        for (unsigned int w = 1; w < 4; w++) {
            idx = idx_zplane + w * x_stride * y_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += 4 * x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
        }

        // u=0,4 v=1-3 w=1-3
        for (unsigned int w = 1; w < 4; w++) {
            idx = idx_zplane + w * x_stride * y_stride;
            idx += x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
        }

        if (sum_idx > 0) {
            interpolated_values[i] = sum / double(sum_idx);
        } else {
            interpolated_values[i] = 0.0;
        }
    }
    return;
}


Sift::Sift(
        const unsigned int x_size,
        const unsigned int y_size,
        const unsigned int z_size,
        const unsigned int x_sub_size,
        const unsigned int y_sub_size,
        const unsigned int dx,
        const unsigned int dy,
        const unsigned int dw,
        const unsigned int num_gpus,
        const unsigned int num_streams)
    : x_size_(x_size), y_size_(y_size), z_size_(z_size),
        x_sub_size_(x_sub_size), y_sub_size_(y_sub_size),
        dx_(dx), dy_(dy), dw_(dw),
        num_gpus_(num_gpus), num_streams_(num_streams),
        subdom_data_(num_gpus) {

    logger_ = spdlog::get("console");
    if (! logger_) {
        logger_ = spdlog::stdout_logger_mt("console");
    }
#ifdef DEBUG_OUTPUT
    spdlog::set_level(spdlog::level::debug);
#else
    spdlog::set_level(spdlog::level::info);
#endif

    size_t log_q_size = 4096;
    spdlog::set_async_mode(log_q_size);

    num_x_sub_ = get_num_blocks(x_size_, x_sub_size_);
    num_y_sub_ = get_num_blocks(y_size_, y_sub_size_);

    x_sub_stride_ = x_sub_size_ + 2 * dw_;
    y_sub_stride_ = y_sub_size_ + 2 * dw_;

    dx_stride_ = dx_ + 2 * dw_;
    dy_stride_ = dy_ + 2 * dw_;
    z_stride_ = z_size_ + 2 * dw_;
#ifdef DEBUG_OUTPUT
    logger_->info("x_size={}, x_sub_size={}, num_x_sub={}, x_sub_stride={}, dx={}, dx_stride={}",
            x_size_, x_sub_size_, num_x_sub_, x_sub_stride_, dx_, dx_stride_);
    logger_->info("y_size={}, y_sub_size={}, num_y_sub={}, y_sub_stride={}, dy={}, dy_stride={}",
            y_size_, y_sub_size_, num_y_sub_, y_sub_stride_, dy_, dy_stride_);
    logger_->info("z_size={}, dw={}, z_stride={}", z_size_, dw_, z_stride_);
#endif


    dom_data_ = std::make_shared<DomainDataOnHost>(x_size_, y_size_, z_size_);

    for (unsigned int i = 0; i < num_gpus_; i++) {
        hipSetDevice(i);

        subdom_data_[i] = std::make_shared<SubDomainDataOnGPU>(x_sub_stride_, y_sub_stride_, z_stride_, num_streams_);

        for (unsigned int j = 0; j < num_streams_; j++) {
            subdom_data_[i]->stream_data[j] = std::make_shared<SubDomainDataOnStream>(dx_stride_, dy_stride_, z_stride_);

            hipStreamCreate(&subdom_data_[i]->stream_data[j]->stream);
        }
    }
    hipSetDevice(0);


    unsigned int idx_gpu = 0;
    for (unsigned int y_sub_i = 0; y_sub_i < num_y_sub_; y_sub_i++) {
        for (unsigned int x_sub_i = 0; x_sub_i < num_x_sub_; x_sub_i++) {
            subdom_data_[idx_gpu]->x_sub_i_list.push_back(x_sub_i);
            subdom_data_[idx_gpu]->y_sub_i_list.push_back(y_sub_i);

            idx_gpu++;
            if (idx_gpu == num_gpus) {
                idx_gpu = 0;
            }
        }
    }

}

Sift::~Sift() {
    for (unsigned int i = 0; i < num_gpus_; i++) {
        for (unsigned int j = 0; j < num_streams_; j++) {
            hipStreamDestroy(subdom_data_[i]->stream_data[j]->stream);
        }
    }

    //logger_->flush();
}

void Sift::setImage(const double *img)
{
    thrust::copy(img, img + (x_size_ * y_size_ * z_size_), dom_data_->h_image);
}

void Sift::setImage(const std::vector<double>& img)
{
    assert((x_size_ * y_size_ * z_size_) == img.size());

    thrust::copy(img.begin(), img.end(), dom_data_->h_image);
}

void Sift::setMapToBeInterpolated(const int8_t *map)
{
    thrust::copy(map, map + (x_size_ * y_size_ * z_size_), dom_data_->h_map);
}

void Sift::setMapToBeInterpolated(const std::vector<int8_t>& map)
{
    assert((x_size_ * y_size_ * z_size_) == map.size());

    thrust::copy(map.begin(), map.end(), dom_data_->h_map);
}

void Sift::getImage(double *img)
{
    thrust::copy(dom_data_->h_image, dom_data_->h_image + x_size_ * y_size_ * z_size_, img);
}

void Sift::getImage(std::vector<double>& img)
{
    thrust::copy(dom_data_->h_image, dom_data_->h_image + x_size_ * y_size_ * z_size_, img.begin());
}


int Sift::getNumOfGPUTasks(const int gpu_id) {
    return subdom_data_[gpu_id]->x_sub_i_list.size();
}

int Sift::getNumOfStreamTasks(
        const int gpu_id,
        const int stream_id) {
    return 1;
}

void Sift::runOnGPU(
        const int gpu_id,
        const unsigned int gpu_task_id) {

    hipSetDevice(gpu_id);

    std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];
    std::shared_ptr<SubDomainDataOnStream> stream_data0 = subdom_data->stream_data[0];

    unsigned int x_sub_i = subdom_data->x_sub_i_list[gpu_task_id];
    unsigned int y_sub_i = subdom_data->y_sub_i_list[gpu_task_id];
#ifdef DEBUG_OUTPUT
    CudaTimer timer;
    logger_->info("===== gpu_id={} x_sub_i={} y_sub_i={}", gpu_id, x_sub_i, y_sub_i);
#endif

    unsigned int x_sub_start = x_sub_i * x_sub_size_;
    unsigned int x_sub_delta = get_delta(x_size_, x_sub_i, x_sub_size_);
    unsigned int y_sub_start = y_sub_i * y_sub_size_;
    unsigned int y_sub_delta = get_delta(y_size_, y_sub_i, y_sub_size_);
    unsigned int base_x_sub  = (x_sub_i > 0 ? 0 : dw_);
    unsigned int base_y_sub  = (y_sub_i > 0 ? 0 : dw_);

    unsigned int padding_x_sub_start = x_sub_start - (x_sub_i > 0 ? dw_ : 0);
    unsigned int padding_x_sub_delta = x_sub_delta + (x_sub_i > 0 ? dw_ : 0) + (x_sub_i < num_x_sub_ - 1 ? dw_ : 0);
    unsigned int padding_y_sub_start = y_sub_start - (y_sub_i > 0 ? dw_ : 0);
    unsigned int padding_y_sub_delta = y_sub_delta + (y_sub_i > 0 ? dw_ : 0) + (y_sub_i < num_y_sub_ - 1 ? dw_ : 0);
#ifdef DEBUG_OUTPUT
    unsigned int x_sub_end = x_sub_start + x_sub_delta;
    unsigned int y_sub_end = y_sub_start + y_sub_delta;
    logger_->debug("x_sub=({},{},{}) y_sub=({},{},{})", x_sub_start, x_sub_delta, x_sub_end, y_sub_start, y_sub_delta, y_sub_end);
    logger_->debug("base_x_sub={},base_y_sub={}", base_x_sub, base_y_sub);
#endif

    size_t padded_sub_volume_size = x_sub_stride_ * y_sub_stride_ * z_stride_;

    int8_t *padded_sub_map;
    double *padded_sub_image;
    hipHostAlloc(&padded_sub_map,   padded_sub_volume_size * sizeof(int8_t), hipHostMallocPortable);
    hipHostAlloc(&padded_sub_image, padded_sub_volume_size * sizeof(double), hipHostMallocPortable);

    // First set all values to -1
    thrust::fill(padded_sub_map, padded_sub_map + padded_sub_volume_size, -1);

    for (unsigned int k = 0; k < z_size_; k++) {
        for (unsigned int j = 0; j < padding_y_sub_delta; j++) {
            size_t src_idx = dom_data_->sub2ind(padding_x_sub_start, padding_y_sub_start + j, k);
            size_t dst_idx = subdom_data->pad_sub2ind(base_x_sub, base_y_sub + j, dw_ + k);

            int8_t* src_map_begin = &(dom_data_->h_map[src_idx]);
            int8_t* dst_map_begin = &(padded_sub_map[dst_idx]);
            thrust::copy(src_map_begin, src_map_begin + padding_x_sub_delta, dst_map_begin);

            double* src_image_begin = &(dom_data_->h_image[src_idx]);
            double* dst_image_begin = &(padded_sub_image[dst_idx]);
            thrust::copy(src_image_begin, src_image_begin + padding_x_sub_delta, dst_image_begin);
        }
    }

    thrust::fill(thrust::device, subdom_data->padded_image, subdom_data->padded_image + padded_sub_volume_size, 0.0);

    hipMemcpyAsync(
            subdom_data->padded_image,
            padded_sub_image,
            padded_sub_volume_size * sizeof(double),
            hipMemcpyHostToDevice, stream_data0->stream);

#ifdef DEBUG_OUTPUT
    hipStreamSynchronize(stream_data0->stream);
    logger_->info("transfer image data {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
    logger_->info("===== dev image");
    print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_image);
    print_matrix3d_dev(logger_, x_sub_stride_, y_sub_stride_, z_stride_, 0, 0, 0, x_sub_stride_, y_sub_stride_, z_stride_, subdom_data->padded_image);
#endif

    timer.reset();
#endif

    hipMemcpyAsync(
            subdom_data->padded_map,
            padded_sub_map,
            padded_sub_volume_size * sizeof(int8_t),
            hipMemcpyHostToDevice, stream_data0->stream);

#ifdef DEBUG_OUTPUT
    hipStreamSynchronize(stream_data0->stream);
    logger_->info("transfer map data {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
    logger_->debug("===== dev map");
    print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_map);
    print_matrix3d_dev(logger_, x_sub_stride_, y_sub_stride_, z_stride_, 0, 0, 0, x_sub_stride_, y_sub_stride_, z_stride_, subdom_data->padded_map);
#endif

    timer.reset();
#endif

    // clear previous result to zero
    thrust::fill(thrust::device, subdom_data->padded_map_idx, subdom_data->padded_map_idx + padded_sub_volume_size, 0.0);

    auto end_itr = thrust::copy_if(
            thrust::device,
            thrust::make_counting_iterator<unsigned int>(0),
            thrust::make_counting_iterator<unsigned int>(padded_sub_volume_size),
            subdom_data->padded_map,
            subdom_data->padded_map_idx,
            thrust::logical_not<int8_t>());

    subdom_data->padded_map_idx_size = end_itr - subdom_data->padded_map_idx;

    // set all padded map boundaries to 0 for correctness to
    // distinguish boundaries
    thrust::replace(thrust::device, subdom_data->padded_map, subdom_data->padded_map + padded_sub_volume_size, -1, 0);

#ifdef DEBUG_OUTPUT
    hipStreamSynchronize(stream_data0->stream);
    logger_->info("calculate map idx {}", timer.get_laptime());

    logger_->info("padded_map_idx_size={}", subdom_data->padded_map_idx_size);
//    logger_->debug("===== padded_map idx");
//    std::copy(subdom_data->padded_map_idx.begin(), end_itr, std::ostream_iterator<unsigned int>(std::cout, ","));
//    std::cout << std::endl;

    timer.reset();
#endif


    // Each GPU each subdom_data
    // this set the dx and dy start idx for each stream
    unsigned int num_dx = get_num_blocks(x_sub_delta, dx_);
    unsigned int num_dy = get_num_blocks(y_sub_delta, dy_);
    unsigned int stream_id = 0;
    for (unsigned int dy_i = 0; dy_i < num_dy; dy_i++) {
        for (unsigned int dx_i = 0; dx_i < num_dx; dx_i++) {
            subdom_data->stream_data[stream_id]->dx_i_list.push_back(dx_i);
            subdom_data->stream_data[stream_id]->dy_i_list.push_back(dy_i);

            stream_id++;
            if (stream_id == num_streams_) {
                stream_id = 0;
            }
        }
    }
    hipStreamSynchronize(stream_data0->stream);

    hipHostFree(padded_sub_map);
    hipHostFree(padded_sub_image);
}

void Sift::runOnStream(
        const int gpu_id,
        const int stream_id,
        const unsigned int gpu_task_id) {

    hipSetDevice(gpu_id);

    std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];
    std::shared_ptr<SubDomainDataOnStream> stream_data = subdom_data->stream_data[stream_id];

    unsigned int x_sub_i = subdom_data->x_sub_i_list[gpu_task_id];
    unsigned int y_sub_i = subdom_data->y_sub_i_list[gpu_task_id];
    unsigned int x_sub_delta = get_delta(x_size_, x_sub_i, x_sub_size_);
    unsigned int y_sub_delta = get_delta(y_size_, y_sub_i, y_sub_size_);

#ifdef DEBUG_OUTPUT
    CudaTimer timer(stream_data->stream);
#endif

    for (auto dx_itr = stream_data->dx_i_list.begin(), dy_itr = stream_data->dy_i_list.begin();
            dx_itr != stream_data->dx_i_list.end() || dy_itr != stream_data->dy_i_list.end(); dx_itr++, dy_itr++) {

        unsigned int dx_i = *dx_itr;
        unsigned int dy_i = *dy_itr;

        unsigned int dx_start = dx_i * dx_;
        unsigned int dx_delta = get_delta(x_sub_delta, dx_i, dx_);
        unsigned int dx_end   = dx_start + dx_delta;
        unsigned int dy_start = dy_i * dy_;
        unsigned int dy_delta = get_delta(y_sub_delta, dy_i, dy_);
        unsigned int dy_end   = dy_start + dy_delta;

#ifdef DEBUG_OUTPUT
        logger_->info("dx_i={}, dy_i={}", dx_i, dy_i);
        logger_->info("x=({},{},{}) y=({},{},{}), dw={}", dx_start, dx_delta, dx_end, dy_start, dy_delta, dy_end, dw_);
        logger_->info("padded_map_idx_size={}", subdom_data->padded_map_idx_size);
#endif


        // for each subdomain of stream of this for loop
        unsigned int *padded_map_idx;
        hipMalloc(&padded_map_idx, subdom_data->padded_map_idx_size * sizeof(unsigned int));

        RangeCheck range_check { x_sub_stride_, y_sub_stride_,
            dx_start + dw_, dx_end + dw_, dy_start + dw_, dy_end + dw_, dw_, z_size_ + dw_ };

        // copy the relevant (in range) idx elements from the
        // global GPU padded_map_idx to the local padded_map_idx for each
        // sub stream (subdom_data->padded_map_idx[stream_id])
        auto end_itr = thrust::copy_if(
                thrust::device,
                subdom_data->padded_map_idx,
                subdom_data->padded_map_idx + subdom_data->padded_map_idx_size,
                padded_map_idx,
                range_check);

        unsigned int padded_map_idx_size = end_itr - padded_map_idx;

#ifdef DEBUG_OUTPUT
        logger_->info("padded_map_idx_size={}", padded_map_idx_size);
        logger_->info("transfer map idx {}", timer.get_laptime());

        hipStreamSynchronize(stream_data->stream);

        thrust::device_vector<unsigned int> dbg_d_padded_map_idx(padded_map_idx, padded_map_idx + padded_map_idx_size);
        thrust::host_vector<unsigned int> dbg_h_padded_map_idx(dbg_d_padded_map_idx);
        for (unsigned int i = 0; i < padded_map_idx_size; i++) {
            logger_->debug("padded_map_idx={}", dbg_h_padded_map_idx[i]);
        }
        timer.reset();
#endif
        if (padded_map_idx_size == 0) {
#ifdef DEBUG_OUTPUT
            logger_->debug("no map to be padded");
#endif
            continue;
        }

        double *interpolated_values;
        hipMalloc(&interpolated_values, padded_map_idx_size * sizeof(double));


        unsigned int num_blocks = get_num_blocks(padded_map_idx_size, 1024);
#ifdef DEBUG_OUTPUT
        logger_->info("num_blocks={}", num_blocks);
#endif

        interpolate_volumes<<<num_blocks, 1024, 0, stream_data->stream>>>(
                x_sub_stride_, y_sub_stride_, padded_map_idx_size,
                padded_map_idx,//substream map
                subdom_data->padded_map,//global map for GPU
                subdom_data->padded_image,
                interpolated_values);

#ifdef DEBUG_OUTPUT
        logger_->info("interpolate volumes {}", timer.get_laptime());

        //debug
//        hipStreamSynchronize(stream_data->stream);
//        std::copy(interpolated_values.begin(),
//                  interpolated_values.begin() + padded_map_idx_size,
//                  std::ostream_iterator<double>(std::cout, ","));
//        std::cout << std::endl;

        timer.reset();
#endif

        double *h_interpolated_values;
        hipHostAlloc(&h_interpolated_values, padded_map_idx_size * sizeof(double), hipHostMallocPortable);

        hipMemcpyAsync(
                h_interpolated_values,
                interpolated_values,
                padded_map_idx_size * sizeof(double),
                hipMemcpyDeviceToHost, stream_data->stream);

        unsigned int *h_padded_map_idx;
        hipHostAlloc(&h_padded_map_idx, padded_map_idx_size * sizeof(unsigned int), hipHostMallocPortable);

        hipMemcpyAsync(
                h_padded_map_idx,
                padded_map_idx,
                padded_map_idx_size * sizeof(unsigned int),
                hipMemcpyDeviceToHost, stream_data->stream);

        hipStreamSynchronize(stream_data->stream);
        for (unsigned int i = 0; i < padded_map_idx_size; i++) {
            unsigned int padding_x;
            unsigned int padding_y;
            unsigned int padding_z;
            ind2sub(x_sub_stride_, y_sub_stride_, h_padded_map_idx[i], padding_x, padding_y, padding_z);
            size_t idx = dom_data_->sub2ind(padding_x - dw_, padding_y - dw_, padding_z - dw_);

            dom_data_->h_image[idx] = h_interpolated_values[i];
        }

        hipFree(padded_map_idx);
        hipFree(interpolated_values);

        hipHostFree(h_interpolated_values);
        hipHostFree(h_padded_map_idx);

#ifdef DEBUG_OUTPUT
        logger_->info("transfer d2h and copy interpolated values {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
        logger_->debug("===== host interp image");
        print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_image);
#endif
#endif
    }
}


} // namespace cudautils

