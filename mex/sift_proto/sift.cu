#include "hip/hip_runtime.h"
#include <iostream>
#include <future>

#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>

#include <hip/hip_runtime.h>
/*#include <hip/hip_runtime_api.h>*/
#include <cmath>
/*#include <numeric> //std::inner_product*/

#include "sift.h"
#include "matrix_helper.h"
#include "cuda_timer.h"

#include "spdlog/spdlog.h"

namespace cudautils {

struct isnan_test {
    __host__ __device__ bool operator() (const float a) const {
        return isnan(a);
    }
};

// row major order index into the descriptor vector
// note the descriptor vector length is determined by 
// sift_params.IndexSize ^ 3 * sift_params.nFaces
// this is why i, j, and k are dimensions of stride sift_params.IndexSize
__device__ __host__
int bin_sub2ind_row(int i, int j, int k, uint16_t l, const cudautils::SiftParams sift_params) {
    return (int) l + sift_params.nFaces * (k + j * pow(sift_params.IndexSize, 1) + i
            * pow(sift_params.IndexSize, 2));
}

// column major order index into the descriptor vector
// note the descriptor vector length is determined by 
// sift_params.IndexSize ^ 3 * sift_params.nFaces
// this is why i, j, and k are dimensions of stride sift_params.IndexSize
__device__ __host__
int bin_sub2ind(int i, int j, int k, uint16_t l, const cudautils::SiftParams sift_params) {
    return (int) i + j * sift_params.IndexSize + k * pow(sift_params.IndexSize, 2)
        + l * pow(sift_params.IndexSize, 3);
}

__device__ __host__
void place_in_index(double* index, double mag, int i, int j, int k, 
        double* yy, uint16_t* ix, const cudautils::SiftParams sift_params) {

    double tmpsum = 0.0;
    int bin_index;
    if (sift_params.Smooth_Flag) {
        for (int tessel=0; tessel < sift_params.Tessel_thresh; tessel++) {
            tmpsum += pow(yy[tessel], sift_params.Smooth_Var);
        }

        // Add three nearest tesselation faces
        for (int ii=0; ii < sift_params.Tessel_thresh; ii++) {
            bin_index = bin_sub2ind(i, j, k, ix[ii], sift_params);

/*#ifdef DEBUG_OUTPUT*/
            /*printf("i%d j%d k%d ix[ii]%d bin_index%d yy[ii]%f, index+=%.3f\n", i, j, k,*/
                    /*ix[ii], bin_index, yy[ii], mag * pow(yy[ii], sift_params.Smooth_Var ) / tmpsum);*/
/*#endif*/
            index[bin_index] +=  mag * pow(yy[ii], sift_params.Smooth_Var ) / tmpsum;
        }
    } else {
        bin_index = bin_sub2ind(i, j, k, ix[0], sift_params);
        index[bin_index] += mag;
    }
    return;
}

// matrix multiply in row memory order 
// first is a matrix in row order
// second is the array multiply
// assumes length of second = cols of first
__device__ __host__
void dot_product(double* first, double* second, double* out, int rows,
        int cols) {
    for (int i=0; i < rows; i++) {
        double sum = 0.0;
        for (int j=0; j < cols; j++) {
            sum += first[j + i * cols] * second[j];
        }
        out[i] = sum;
    }
}


// matrix multiply in col memory order 
// first is a matrix in column order
// second is the array multiply
// assumes length of second = cols of first
__device__ __host__
void dot_product_col_ord(double* first, double* second, double* out, int rows,
        int cols) {
    for (int i=0; i < rows; i++) {
        double sum = 0.0;
        for (int j=0; j < cols; j++) {
            sum += first[i + j * rows] * second[j];
        }
        out[i] = sum;
    }
}

/*void gpu_blas_mmul(hipblasHandle_t &handle, const double* A, const double* B,*/
        /*double* C, const int m, const int k, const int n) {*/
    /*const float alf = 1;*/
    /*const float bet = 0;*/
/*}*/

__global__
void get_grad_ori_vector_wrapper(double* image, unsigned long long idx, unsigned int
        x_stride, unsigned int y_stride, double vect[3], double* yy, uint16_t* ix,
        const cudautils::SiftParams sift_params, double* device_centers, double* mag) {

    *mag = cudautils::get_grad_ori_vector(thrust::raw_pointer_cast(&image[0]), 
        idx, x_stride, y_stride, thrust::raw_pointer_cast(&vect[0]),
        thrust::raw_pointer_cast(&yy[0]), thrust::raw_pointer_cast(&ix[0]),
        sift_params, thrust::raw_pointer_cast(&device_centers[0]));
    return;
}

// assumes r,c,s lie within accessible image boundaries
__device__ 
double get_grad_ori_vector(double* image, unsigned long long idx, unsigned int
        x_stride, unsigned int y_stride, double vect[3], double* yy, uint16_t* ix,
        const cudautils::SiftParams sift_params, double* device_centers) {


    /* this is literal translation from Scovanner et al. 3DSIFT, 
       even though it seems xgrad and ygrad are switched, and ygrad seems to be
       in wrong direction
    */
    double xgrad = image[idx + x_stride] - image[idx - x_stride];
    double ygrad = image[idx - 1] - image[idx + 1];
    double zgrad = image[idx + x_stride * y_stride] - image[idx - x_stride * y_stride];

    /*printf("ggov idx%u image[idx -1] %f image[idx+1] %f\n\txgrad %f y %f z %f\n",*/
            /*idx, image[idx - 1], image[idx + 1], xgrad, ygrad, zgrad);*/

    double mag = sqrt(xgrad * xgrad + ygrad * ygrad + zgrad * zgrad);

    xgrad /= mag;
    ygrad /= mag;
    zgrad /= mag;

    if (mag != 0.0) {
        vect[0] = xgrad;
        vect[1] = ygrad;
        vect[2] = zgrad;
    } 

    //Find the nearest tesselation face indices
    int dims = 3;
    // N = sift_params.nFaces 
    int N = sift_params.fv_centers_len / dims;
    dot_product(device_centers, vect, yy, N, dims);

    /*int di;*/
    /*printf("device_centers:\n");*/
    /*for (int i=0; i < N; i++) {*/
        /*for (int j=0; j < dims; j++) {*/
            /*di = j + dims * i;*/
            /*printf("[%d]=%.3f, ", di, device_centers[di]);*/
        /*}*/
        /*printf("\n");*/
    /*}*/

    /*for (int i=0; i < sift_params.nFaces; i++) {*/
        /*[>if (yy[i] != 0.0) {<]*/
        /*[>}<]*/
        /*printf("yy[%d]=%.3f\n", i, yy[i]);*/
        /*if (i < dims)*/
            /*printf("vect[%d]=%f\n", i, vect[i]);*/
    /*}*/
    /*printf("sort yy\n");*/
    
    // overwrite idxs 1 : N, N can not exceed the length of ori_hist
    thrust::sequence(thrust::device, ix, ix + sift_params.nFaces);
    // descending order by ori_hist
    thrust::sort_by_key(thrust::device, yy, yy + sift_params.nFaces, ix, thrust::greater<double>());

    /*for (int i=0; i < sift_params.nFaces; i++) {*/
        /*[>if (yy[i] != 0.0) {<]*/
        /*[>}<]*/
        /*printf("yy[%d]=%.3f\n", i, yy[i]);*/
        /*printf("ix[%d]=%d\n", i, ix[i]);*/
    /*}*/

    return mag;
}

/*r, c, s is the pixel index (x, y, z dimensions respect.) in the image within the radius of the */
/*keypoint before clamped*/
/*For each pixel, take a neighborhhod of xyradius and tiradius,*/
/*bin it down to the sift_params.IndexSize dimensions*/
/*thus, i_indx, j_indx, s_indx represent the binned index within the radius of the keypoint*/
__device__
void add_sample(double* index, double* image, double distsq, unsigned long long
        idx, unsigned int x_stride, unsigned int y_stride, int i_bin, int j_bin, int k_bin, 
        const cudautils::SiftParams sift_params, double* device_centers,
        uint16_t* ix, double* yy) {

    double sigma = sift_params.SigmaScaled;
    double weight = exp(-(distsq / (2.0 * sigma * sigma)));

    double vect[3] = {1.0, 0.0, 0.0};

    // gradient and orientation vectors calculated from 3D halo/neighboring
    // pixels
    double mag = get_grad_ori_vector(image, idx, x_stride, y_stride, vect, yy, ix, sift_params, 
            device_centers);
    mag *= weight; // scale magnitude by gaussian 

    place_in_index(index, mag, i_bin, j_bin, k_bin, yy, ix, sift_params);
    return;
}


// floor quotient, add 1
// clamp bin idx to IndexSize
__device__ __host__
int get_bin_idx(int orig, int radius, int IndexSize) {
    int idx = (int) floor((orig + radius) / (2.0 * (double) radius / IndexSize));
    if (idx >= IndexSize) // clamp to IndexSize
        idx = IndexSize - 1;
    return idx;
}

__device__
double* key_sample(const cudautils::SiftParams sift_params, 
        cudautils::Keypoint key, double* image, unsigned long long idx,
        unsigned int x_stride, unsigned int y_stride, 
        double* device_centers, uint16_t* ix, double* yy,
        double* index) {

    double xySpacing = (double) sift_params.xyScale * sift_params.MagFactor;
    double tSpacing = (double) sift_params.tScale * sift_params.MagFactor;

    int xyiradius = rint(1.414 * xySpacing * (sift_params.IndexSize + 1) / 2.0);
    int tiradius = rint(1.414 * tSpacing * (sift_params.IndexSize + 1) / 2.0);

    printf("xyiradius %d, tiradius %d\n", xyiradius, tiradius);
    printf("x %d, y %d, z %d\n", key.x, key.y, key.z);

    // Surrounding radius of pixels are binned for computation 
    // according to sift_params.IndexSize
    int r, c, t, i_bin, j_bin, k_bin;
    double distsq;
    int counter = 0;
    int inner_counter = 0;
    unsigned long long update_idx;
    for (int i = -xyiradius; i <= xyiradius; i++) {
        for (int j = -xyiradius; j <= xyiradius; j++) {
            for (int k = -tiradius; k <= tiradius; k++) {
                counter++;

                // FIXME check for CUDA pow function
                distsq = (double) pow(i,2) + pow(j,2) + pow(k,2);

                // Find bin idx
                // FIXME check correct
                i_bin = get_bin_idx(i, xyiradius, sift_params.IndexSize);
                j_bin = get_bin_idx(j, xyiradius, sift_params.IndexSize);
                k_bin = get_bin_idx(k, tiradius, sift_params.IndexSize);
                
                // Find original image pixel idx
                r = key.x + i;
                c = key.y + j;
                t = key.z + k;
                printf("r %d, c %d, t %d\n", r, c, t);

                // FIXME does this collide with GPU splitting?
                // only add if within image range
                if (!(r < 0  ||  r >= sift_params.image_size0 ||
                        c < 0  ||  c >= sift_params.image_size1
                        || t < 0 || t >= sift_params.image_size2)) {

                    inner_counter++;
                    // image is assumed as column order
                    // make sure it isn't cast to unsigned
                    update_idx = (long long) idx + i + (int) x_stride * j +
                        (int) x_stride * (int) y_stride * k;
                    add_sample(index, image, distsq, update_idx, x_stride, y_stride,
                            i_bin, j_bin, k_bin, sift_params,
                            device_centers, ix, yy);
                }
            }
        }
    }
    printf("counter%d\n",counter);
    printf("inner_counter%d\n",inner_counter);

    return index;
}

__device__
double* build_ori_hists(int x, int y, int z, unsigned long long idx, unsigned int
        x_stride, unsigned int y_stride, int radius, double* image, 
        const cudautils::SiftParams sift_params, double* device_centers,
        uint16_t* ix, double* yy, double* ori_hist) {

    double mag;
    double vect[3] = {1.0, 0.0, 0.0};

    int r, c, t;
    for (int i = -radius; i <= radius; i++) {
        for (int j = -radius; j <= radius; j++) {
            for (int k = -radius; k <= radius; k++) {
                // Find original image pixel idx
                r = x + i;
                c = y + j;
                t = z + k;

                // only add if within image range
                if (!(r < 0  ||  r >= sift_params.image_size0 ||
                        c < 0  ||  c >= sift_params.image_size1
                        || t < 0 || t >= sift_params.image_size2)) {
                    /*gradient and orientation vectors calculated from 3D halo/neighboring pixels*/
                    mag = get_grad_ori_vector(image, idx, x_stride, y_stride,
                            vect, yy, ix, sift_params, device_centers);
                    ori_hist[ix[0]] += mag;
                }
            }
        }
    }
    return ori_hist;
}

__device__
void normalize_arr(double* arr, int len) {

    double sqlen = 0.0;
    for (int i=0; i < len; i++) {
        sqlen += arr[i] * arr[i];
    }

    double fac = 1.0 / sqrt(sqlen);
    for (int i=0; i < len; i++) {
        arr[i] = arr[i] * fac;
    }
    return;
}

__device__
cudautils::Keypoint make_keypoint_sample(cudautils::Keypoint key, double*
        image, const cudautils::SiftParams sift_params, unsigned int thread_idx, unsigned long long idx,
        unsigned int x_stride, unsigned int y_stride, double * descriptors,
        double* device_centers, uint16_t* ix, double* yy) {

    bool changed = false;

    // default N=640; 5120 bytes
    int N = sift_params.descriptor_len;
    double* index = &(descriptors[thread_idx * sift_params.descriptor_len]);
    memset(index, 0.0, N * sizeof(double));

    //FIXME make sure is in column order
    key_sample(sift_params, key, image, idx, x_stride, y_stride,
            device_centers, ix, yy, index);

    /*for (int i=0; i < sift_params.descriptor_len; i++) {*/
        /*if (index[i] != 0) */
            /*printf("index[%d]=%.4f, ",i, index[i]);*/
    /*}*/
    /*printf("\n");*/

    normalize_arr(index, N);

    for (int i=0; i < N; i++) {
        if (index[i] > sift_params.MaxIndexVal) {
            index[i] = sift_params.MaxIndexVal;
            changed = true;
        }
    }

    if (changed) {
        normalize_arr(index, N);
    }

    int intval;
    for (int i=0; i < N; i++) {
        intval = rint(512.0 * index[i]);
        //FIXME cuda function for min?
        index[i] =  (double) min(255, intval);
    }
    return key;
}

__device__
cudautils::Keypoint make_keypoint(double* image, int x, int y, int z,
        unsigned int thread_idx, unsigned long long idx, unsigned int x_stride, unsigned int y_stride,
        const cudautils::SiftParams sift_params, double * descriptors, double*
        device_centers, uint16_t* ix, double* yy) {
    cudautils::Keypoint key;
    key.x = x;
    key.y = y;
    key.z = z;

    return make_keypoint_sample(key, image, sift_params, thread_idx, idx,
            x_stride, y_stride, descriptors, device_centers, ix, yy);
}

/* Main function of 3DSIFT Program from http://www.cs.ucf.edu/~pscovann/
Inputs:
image - a 3 dimensional matrix of double
xyScale and tScale - affects both the scale and the resolution, these are
usually set to 1 and scaling is done before calling this function
x, y, and z - the location of the center of the keypoint where a descriptor is requested

Outputs:
keypoint - the descriptor, varies in size depending on values in LoadParams.m
reRun - a flag (0 or 1) which is set if the data at (x,y,z) is not
descriptive enough for a good keypoint
*/
__global__
void create_descriptor(
        unsigned int x_stride,
        unsigned int y_stride,
        unsigned int x_sub_start,
        unsigned int y_sub_start,
        unsigned int dw,
        unsigned int map_idx_size,
        unsigned int *map_idx,
        int8_t *map,
        double *image,
        const cudautils::SiftParams sift_params, 
        double* device_centers,
        double *descriptors,
        uint16_t* idx_scratch,
        double* yy_scratch,
        uint16_t* ori_idx_scratch,
        double* ori_scratch) {

    // thread per keypoint in this substream
    unsigned int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= map_idx_size) return;
    // map_idx holds the relevant image idxs only for the substream
    // map_idx_size matchs total # of threads
    // idx describes the linear index for current GPUs section of the image and corresponding map
    unsigned long long idx = map_idx[thread_idx];
    /*printf("create_descriptor image[idx -1] %f image[idx+1] %f\n", image[idx - 1], image[idx + 1]);*/

    // column-major order since image is from matlab
    int x, y, z;
    unsigned int padding_x;
    unsigned int padding_y;
    unsigned int padding_z;
    ind2sub(x_stride, y_stride, idx, padding_x, padding_y, padding_z);
    // correct for dw_ padding, 0-indexed for checking boundaries
    x = x_sub_start + padding_x - dw;
    y = y_sub_start + padding_y - dw;
    z = padding_z - dw;
    
    uint16_t* ix = (uint16_t*) &(idx_scratch[thread_idx * sift_params.nFaces]);
    cudaCheckPtr(ix);

    double *yy = (double*) &(yy_scratch[thread_idx * sift_params.nFaces]);
    cudaCheckPtr(yy);

    if (sift_params.TwoPeak_Flag) {
        int radius = rint(sift_params.xyScale * 3.0);

        // init ori hist indices
        int ori_hist_len = sift_params.nFaces; //default 80
        uint16_t* ori_hist_idx = &(ori_idx_scratch[ori_hist_len * thread_idx]);
        cudaCheckPtr(ori_hist_idx);
        thrust::sequence(thrust::device, ori_hist_idx, ori_hist_idx + ori_hist_len);

        //init ori histogram
        /*double* ori_hist = (double*) malloc(sift_params.nFaces * sizeof(double));*/
        double* ori_hist = &(ori_scratch[ori_hist_len * thread_idx]);
        cudaCheckPtr(ori_hist);
        memset(ori_hist, 0.0, ori_hist_len * sizeof(double));

        build_ori_hists(x, y, z, idx, x_stride, y_stride, radius, image,
                sift_params, device_centers, ix, yy, ori_hist);
        // descending order according to ori_hist
        thrust::sort_by_key(thrust::device, ori_hist, ori_hist +
                ori_hist_len, ori_hist_idx, thrust::greater<double>());
            
        // FIXME have this in sift_params
        int dims = 3; float thresh = .9;
        double prod01, prod02;
        dot_product(&(device_centers[dims * ori_hist_idx[0]]),
            &(device_centers[dims * ori_hist_idx[1]]), &prod01, 1, dims);
        dot_product(&(device_centers[dims * ori_hist_idx[0]]),
            &(device_centers[dims * ori_hist_idx[2]]), &prod02, 1, dims);
        if ( ( prod01 > thresh) &&
             ( prod02 > thresh) ) {
            // FIXME remove this since memory is never accessed
            /*memset(&(descriptors[idx]), 0, sift_params.descriptor_len * sizeof(double));*/
            // mark this keypoint as null in map
            map_idx[thread_idx] = nan("");
            // FIXME print in final version
            /*printf("Removed keypoint from thread: %u, desc index: %u, x:%d
             * y:%d z:%d\n", thread_idx, idx, x, y, z);*/
            return ;
        }

    }

    cudautils::Keypoint key = make_keypoint(image, x, y, z, thread_idx, idx,
            x_stride, y_stride, sift_params, descriptors, device_centers, ix,
            yy);

    return;
}

/*Define the constructor for the SIFT class*/
/*See the class Sift definition in sift.h*/
Sift::Sift(
        const unsigned int x_size,
        const unsigned int y_size,
        const unsigned int z_size,
        const unsigned int x_sub_size,
        const unsigned int y_sub_size,
        const unsigned int dx,
        const unsigned int dy,
        const unsigned int dw,
        const unsigned int num_gpus,
        const unsigned int num_streams,
        const cudautils::SiftParams sift_params,
        const double* fv_centers)
    : x_size_(x_size), y_size_(y_size), z_size_(z_size),
        x_sub_size_(x_sub_size), y_sub_size_(y_sub_size),
        dx_(dx), dy_(dy), dw_(dw),
        num_gpus_(num_gpus), num_streams_(num_streams),
        sift_params_(sift_params),
        fv_centers_(fv_centers),
        subdom_data_(num_gpus) {

    logger_ = spdlog::get("console");
    if (! logger_) {
        logger_ = spdlog::stdout_logger_mt("console");
    }
#ifdef DEBUG_OUTPUT
    spdlog::set_level(spdlog::level::debug);
#else
    spdlog::set_level(spdlog::level::info);
#endif

    size_t log_q_size = 4096;
    spdlog::set_async_mode(log_q_size);

    num_x_sub_ = get_num_blocks(x_size_, x_sub_size_);
    num_y_sub_ = get_num_blocks(y_size_, y_sub_size_);

    x_sub_stride_ = x_sub_size_ + 2 * dw_;
    y_sub_stride_ = y_sub_size_ + 2 * dw_;

    dx_stride_ = dx_ + 2 * dw_;
    dy_stride_ = dy_ + 2 * dw_;
    z_stride_ = z_size_ + 2 * dw_;
#ifdef DEBUG_OUTPUT
    logger_->info("x_size={}, x_sub_size={}, num_x_sub={}, x_sub_stride={}, dx={}, dx_stride={}",
            x_size_, x_sub_size_, num_x_sub_, x_sub_stride_, dx_, dx_stride_);
    logger_->info("y_size={}, y_sub_size={}, num_y_sub={}, y_sub_stride={}, dy={}, dy_stride={}",
            y_size_, y_sub_size_, num_y_sub_, y_sub_stride_, dy_, dy_stride_);
    logger_->info("z_size={}, dw={}, z_stride={}", z_size_, dw_, z_stride_);
#endif


    dom_data_ = std::make_shared<DomainDataOnHost>(x_size_, y_size_, z_size_);

    for (unsigned int i = 0; i < num_gpus_; i++) {
        hipSetDevice(i);

        subdom_data_[i] = std::make_shared<SubDomainDataOnGPU>(x_sub_stride_, y_sub_stride_, z_stride_, num_streams_);

        for (unsigned int j = 0; j < num_streams_; j++) {
            subdom_data_[i]->stream_data[j] = std::make_shared<SubDomainDataOnStream>(dx_stride_, dy_stride_, z_stride_);

            hipStreamCreate(&subdom_data_[i]->stream_data[j]->stream);
        }
    }
    hipSetDevice(0);


    unsigned int idx_gpu = 0;
    for (unsigned int y_sub_i = 0; y_sub_i < num_y_sub_; y_sub_i++) {
        for (unsigned int x_sub_i = 0; x_sub_i < num_x_sub_; x_sub_i++) {
            subdom_data_[idx_gpu]->x_sub_i_list.push_back(x_sub_i);
            subdom_data_[idx_gpu]->y_sub_i_list.push_back(y_sub_i);

            idx_gpu++;
            if (idx_gpu == num_gpus) {
                idx_gpu = 0;
            }
        }
    }

}

Sift::~Sift() {
    for (unsigned int i = 0; i < num_gpus_; i++) {
        for (unsigned int j = 0; j < num_streams_; j++) {
            hipStreamDestroy(subdom_data_[i]->stream_data[j]->stream);
        }
    }

    //logger_->flush();
}

void Sift::setImage(const double *img)
{
    thrust::copy(img, img + (x_size_ * y_size_ * z_size_), dom_data_->h_image);
}

void Sift::setImage(const std::vector<double>& img)
{
    assert((x_size_ * y_size_ * z_size_) == img.size());

    thrust::copy(img.begin(), img.end(), dom_data_->h_image);

    //FIXME delete this
    /*printf("setImage\n");*/
    /*fflush(stdout);*/
    /*for (int i=0; i < 100; i++) {*/
        /*[>if (dom_data_->h_image[i] != 0.0) {<]*/
        /*printf("setImage h_image[%d]: %f\n", i, dom_data_->h_image[i]);*/
        /*printf("setImage img[%d]: %f\n", i, img[i]);*/
        /*fflush(stdout);*/
        /*[>}<]*/
    /*}*/
}

void Sift::setMapToBeInterpolated(const int8_t *map)
{
    thrust::copy(map, map + (x_size_ * y_size_ * z_size_), dom_data_->h_map);
}

void Sift::setMapToBeInterpolated(const std::vector<int8_t>& map)
{
    assert((x_size_ * y_size_ * z_size_) == map.size());

    thrust::copy(map.begin(), map.end(), dom_data_->h_map);
}

void Sift::getKeystore(cudautils::Keypoint_store *keystore)
{
    keystore->len = dom_data_->keystore->len;
    keystore->buf = (cudautils::Keypoint*) malloc(keystore->len * sizeof(cudautils::Keypoint));
    thrust::copy(dom_data_->keystore->buf, dom_data_->keystore->buf + dom_data_->keystore->len, keystore->buf);
}


void Sift::getImage(double *img)
{
    thrust::copy(dom_data_->h_image, dom_data_->h_image + x_size_ * y_size_ * z_size_, img);
}

void Sift::getImage(std::vector<double>& img)
{
    thrust::copy(dom_data_->h_image, dom_data_->h_image + x_size_ * y_size_ * z_size_, img.begin());
}


int Sift::getNumOfGPUTasks(const int gpu_id) {
    return subdom_data_[gpu_id]->x_sub_i_list.size();
}

int Sift::getNumOfStreamTasks(
        const int gpu_id,
        const int stream_id) {
    return 1;
}

void Sift::runOnGPU(
        const int gpu_id,
        const unsigned int gpu_task_id) {

    cudaSafeCall(hipSetDevice(gpu_id));

    std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];
    std::shared_ptr<SubDomainDataOnStream> stream_data0 = subdom_data->stream_data[0];

    unsigned int x_sub_i = subdom_data->x_sub_i_list[gpu_task_id];
    unsigned int y_sub_i = subdom_data->y_sub_i_list[gpu_task_id];
#ifdef DEBUG_OUTPUT
    CudaTimer timer;
    logger_->info("===== gpu_id={} x_sub_i={} y_sub_i={}", gpu_id, x_sub_i, y_sub_i);
#endif

    unsigned int x_sub_start = x_sub_i * x_sub_size_;
    unsigned int y_sub_start = y_sub_i * y_sub_size_;
    // clamp delta to end value 
    unsigned int x_sub_delta = get_delta(x_size_, x_sub_i, x_sub_size_);
    unsigned int y_sub_delta = get_delta(y_size_, y_sub_i, y_sub_size_);
    // only add in pad factor at first
    unsigned int base_x_sub  = (x_sub_i > 0 ? 0 : dw_);
    unsigned int base_y_sub  = (y_sub_i > 0 ? 0 : dw_);

    // subtract pad factor after first
    unsigned int padding_x_sub_start = x_sub_start - (x_sub_i > 0 ? dw_ : 0);
    unsigned int padding_y_sub_start = y_sub_start - (y_sub_i > 0 ? dw_ : 0);
    unsigned int padding_x_sub_delta = x_sub_delta + (x_sub_i > 0 ? dw_ : 0) + (x_sub_i < num_x_sub_ - 1 ? dw_ : 0);
    unsigned int padding_y_sub_delta = y_sub_delta + (y_sub_i > 0 ? dw_ : 0) + (y_sub_i < num_y_sub_ - 1 ? dw_ : 0);

    // per GPU padded image size
    size_t padded_sub_volume_size = x_sub_stride_ * y_sub_stride_ * z_stride_;

#ifdef DEBUG_OUTPUT
    unsigned int x_sub_end = x_sub_start + x_sub_delta;
    unsigned int y_sub_end = y_sub_start + y_sub_delta;
    logger_->debug("x_sub=({},{},{}) y_sub=({},{},{})", x_sub_start, x_sub_delta, x_sub_end, y_sub_start, y_sub_delta, y_sub_end);
    logger_->debug("base_x_sub={},base_y_sub={}", base_x_sub, base_y_sub);

#ifdef DEBUG_OUTPUT_MATRIX
    // print the x, y, z image / map coordinates of the selected keypoints
    if (gpu_id == 0)  { // don't repeat this for every GPU
        for (long long idx=0; idx < x_size_ * y_size_ * z_size_; idx++) {
            if (! dom_data_->h_map[idx]) {
                unsigned int x;
                unsigned int y;
                unsigned int z;
                ind2sub(x_size_, y_size_, idx, x, y, z);

                logger_->info("h_map 0's: idx={}, x={}, y={}, z={}",
                        idx, x, y, z);
            }
        }
    }
#endif
#endif

    // allocate the per GPU padded map and image
    int8_t *padded_sub_map;
    double *padded_sub_image;
    cudaSafeCall(hipHostAlloc(&padded_sub_map, padded_sub_volume_size *
                sizeof(int8_t), hipHostMallocPortable));
        cudaCheckError();
    cudaSafeCall(hipHostAlloc(&padded_sub_image, padded_sub_volume_size *
                sizeof(double), hipHostMallocPortable));
        cudaCheckError();

    // First set all values to holder value -1
    thrust::fill(padded_sub_map, padded_sub_map + padded_sub_volume_size, -1);

    for (unsigned int k = 0; k < z_size_; k++) {
        for (unsigned int j = 0; j < padding_y_sub_delta; j++) {
            // get row-major / c-order linear index according orig. dim [x_size, y_size, z_size]
            size_t src_idx = dom_data_->sub2ind(padding_x_sub_start, padding_y_sub_start + j, k);
            size_t dst_idx = subdom_data->pad_sub2ind(base_x_sub, base_y_sub + j, dw_ + k);

            int8_t* src_map_begin = &(dom_data_->h_map[src_idx]);
            int8_t* dst_map_begin = &(padded_sub_map[dst_idx]);
            // note this assumes the rows to be contiguous in memory (row-order / c-order)
            thrust::copy(src_map_begin, src_map_begin + padding_x_sub_delta, dst_map_begin);

            double* src_image_begin = &(dom_data_->h_image[src_idx]);
            double* dst_image_begin = &(padded_sub_image[dst_idx]);
            thrust::copy(src_image_begin, src_image_begin + padding_x_sub_delta, dst_image_begin);
        }
    }
    
#ifdef DEBUG_OUTPUT_MATRIX

    //FIXME place this back in DEBUG_OUTPUT_MATRIX above
    // print the x, y, z in padded image / map coordinates of the selected keypoints
    for (long long i=0; i < padded_sub_volume_size; i++) {
        if (!padded_sub_map[i]) {
            unsigned int padding_x;
            unsigned int padding_y;
            unsigned int padding_z;
            ind2sub(x_sub_stride_, y_sub_stride_, i, padding_x, padding_y, padding_z);
            // correct for dw_ padding, matlab is 1-indexed
            unsigned int x = x_sub_start + padding_x - dw_ + 1;
            unsigned int y = y_sub_start + padding_y - dw_ + 1;
            unsigned int z = padding_z - dw_ + 1;

            logger_->info("padded_sub_map 0's (matlab 1-indexed): idx={}, x={}, y={}, z={}",
                    i, x, y, z);
        }
    }

#endif

    thrust::fill(thrust::device, subdom_data->padded_image, subdom_data->padded_image + padded_sub_volume_size, 0.0);

    cudaSafeCall(hipMemcpyAsync(
            subdom_data->padded_image,
            padded_sub_image,
            padded_sub_volume_size * sizeof(double),
            hipMemcpyHostToDevice, stream_data0->stream));

#ifdef DEBUG_OUTPUT
    cudaSafeCall(hipStreamSynchronize(stream_data0->stream));
    logger_->info("transfer image data {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
    logger_->info("===== dev image");
    print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_image);
    print_matrix3d_dev(logger_, x_sub_stride_, y_sub_stride_, z_stride_, 0, 0, 0, x_sub_stride_, y_sub_stride_, z_stride_, subdom_data->padded_image);
#endif

    timer.reset();
#endif

    cudaSafeCall(hipMemcpyAsync(
            subdom_data->padded_map,
            padded_sub_map,
            padded_sub_volume_size * sizeof(int8_t),
            hipMemcpyHostToDevice, stream_data0->stream));

#ifdef DEBUG_OUTPUT
    cudaSafeCall(hipStreamSynchronize(stream_data0->stream));
    logger_->info("transfer map data {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
    logger_->debug("===== dev map");
    print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_map);
    print_matrix3d_dev(logger_, x_sub_stride_, y_sub_stride_, z_stride_, 0, 0, 0, x_sub_stride_, y_sub_stride_, z_stride_, subdom_data->padded_map);
#endif

    timer.reset();
#endif

    // clear previous result to zero
    thrust::fill(thrust::device, subdom_data->padded_map_idx, subdom_data->padded_map_idx + padded_sub_volume_size, 0.0);

    /*Note: padded_sub_volume_size = x_sub_stride_ * y_sub_stride_ * z_stride_;*/
    auto end_itr = thrust::copy_if(
            thrust::device,
            thrust::make_counting_iterator<unsigned int>(0), // count indexes from 0
            thrust::make_counting_iterator<unsigned int>(padded_sub_volume_size), // ...to padded_sub_volume_size
            subdom_data->padded_map, //beginning of stencil sequence
            subdom_data->padded_map_idx, // beginning of sequence to copy into
            thrust::logical_not<int8_t>());//predicate test on every value

    subdom_data->padded_map_idx_size = end_itr - subdom_data->padded_map_idx;

    // set all padded map boundaries (still -1) to 0 for correctness to
    // distinguish boundaries
    thrust::replace(thrust::device, subdom_data->padded_map, subdom_data->padded_map + padded_sub_volume_size, -1, 0);

#ifdef DEBUG_OUTPUT
    cudaSafeCall(hipStreamSynchronize(stream_data0->stream));
    logger_->info("calculate map idx {}", timer.get_laptime());

    logger_->info("padded_map_idx_size={}", subdom_data->padded_map_idx_size);

    timer.reset();
#endif


    // Each GPU each subdom_data
    // this set the dx and dy start idx for each stream
    unsigned int num_dx = get_num_blocks(x_sub_delta, dx_);
    unsigned int num_dy = get_num_blocks(y_sub_delta, dy_);
    unsigned int stream_id = 0;
    for (unsigned int dy_i = 0; dy_i < num_dy; dy_i++) {
        for (unsigned int dx_i = 0; dx_i < num_dx; dx_i++) {
            subdom_data->stream_data[stream_id]->dx_i_list.push_back(dx_i);
            subdom_data->stream_data[stream_id]->dy_i_list.push_back(dy_i);

            stream_id++;
            if (stream_id == num_streams_) {
                stream_id = 0;
            }
        }
    }
    cudaSafeCall(hipStreamSynchronize(stream_data0->stream));

    cudaSafeCall(hipHostFree(padded_sub_map));
    cudaSafeCall(hipHostFree(padded_sub_image));
}

const cudautils::SiftParams Sift::get_sift_params() {
    return sift_params_;
}

void Sift::postrun() {
    // count keypoints
    int total_keypoints = 0;
    for (int gpu_id = 0; gpu_id < num_gpus_; gpu_id++) {
        std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];

        for (int stream_id = 0; stream_id < num_streams_; stream_id++) {
            std::shared_ptr<SubDomainDataOnStream> stream_data =
                subdom_data->stream_data[stream_id];

            total_keypoints += stream_data->keystore->len;
        }
    }

    // allocate for number of keypoints
    dom_data_->keystore->len = total_keypoints;
    hipHostAlloc(&(dom_data_->keystore->buf), dom_data_->keystore->len *
            sizeof(cudautils::Keypoint), hipHostMallocPortable);

    // copy keypoints to host
    int counter = 0;
    for (int gpu_id = 0; gpu_id < num_gpus_; gpu_id++) {
        std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];

        for (int stream_id = 0; stream_id < num_streams_; stream_id++) {
            std::shared_ptr<SubDomainDataOnStream> stream_data =
                subdom_data->stream_data[stream_id];

            for (int i = 0; i < stream_data->keystore->len; i++) {
                dom_data_->keystore->buf[counter] = stream_data->keystore->buf[i];
                counter++;
            }
        }
    }
    return;
}


void Sift::runOnStream(
        const int gpu_id,
        const int stream_id,
        const unsigned int gpu_task_id) {

    hipSetDevice(gpu_id);

    std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];
    std::shared_ptr<SubDomainDataOnStream> stream_data = subdom_data->stream_data[stream_id];

    unsigned int x_sub_i = subdom_data->x_sub_i_list[gpu_task_id];
    unsigned int y_sub_i = subdom_data->y_sub_i_list[gpu_task_id];
    unsigned int x_sub_delta = get_delta(x_size_, x_sub_i, x_sub_size_);
    unsigned int y_sub_delta = get_delta(y_size_, y_sub_i, y_sub_size_);
    unsigned int x_sub_start = x_sub_i * x_sub_size_;
    unsigned int y_sub_start = y_sub_i * y_sub_size_;

#ifdef DEBUG_OUTPUT
    CudaTimer timer(stream_data->stream);
#endif

    // each stream has a individual subsections of data, that each kernel call will operate on
    // these subsections start/stop idx are determined by dx_i and dy_i lists
    for (auto dx_itr = stream_data->dx_i_list.begin(), dy_itr = stream_data->dy_i_list.begin();
            dx_itr != stream_data->dx_i_list.end() || dy_itr != stream_data->dy_i_list.end();
            dx_itr++, dy_itr++) {

        unsigned int dx_i = *dx_itr;
        unsigned int dy_i = *dy_itr;

        unsigned int dx_start = dx_i * dx_;
        unsigned int dx_delta = get_delta(x_sub_delta, dx_i, dx_);
        unsigned int dx_end   = dx_start + dx_delta;
        unsigned int dy_start = dy_i * dy_;
        unsigned int dy_delta = get_delta(y_sub_delta, dy_i, dy_);
        unsigned int dy_end   = dy_start + dy_delta;

#ifdef DEBUG_OUTPUT
        logger_->info("dx_i={}, dy_i={}", dx_i, dy_i);
        logger_->info("x=({},{},{}) y=({},{},{}), dw={}", dx_start, dx_delta, dx_end, dy_start, dy_delta, dy_end, dw_);
        logger_->info("subdom_data->padded_map_idx_size={}", subdom_data->padded_map_idx_size);
#endif

        // create each substream data on device
        unsigned int *substream_padded_map_idx;
        cudaSafeCall(hipMalloc(&substream_padded_map_idx,
                    subdom_data->padded_map_idx_size * sizeof(unsigned int)));

        RangeCheck range_check { x_sub_stride_, y_sub_stride_,
            dx_start + dw_, dx_end + dw_, dy_start + dw_, dy_end + dw_, dw_, z_size_ + dw_ };

        // copy the relevant (in range) idx elements from the
        // global GPU padded_map_idx to the local substream_padded_map_idx 
        auto end_itr = thrust::copy_if(
                thrust::device,
                subdom_data->padded_map_idx,
                subdom_data->padded_map_idx + subdom_data->padded_map_idx_size,
                substream_padded_map_idx,
                range_check);

        unsigned int substream_padded_map_idx_size = end_itr - substream_padded_map_idx;

#ifdef DEBUG_OUTPUT
        logger_->info("substream_padded_map_idx_size={}", substream_padded_map_idx_size);
        logger_->info("transfer map idx {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
        cudaSafeCall(hipStreamSynchronize(stream_data->stream));
        thrust::device_vector<unsigned int> dbg_d_padded_map_idx(substream_padded_map_idx,
                substream_padded_map_idx + substream_padded_map_idx_size);
        thrust::host_vector<unsigned int> dbg_h_padded_map_idx(dbg_d_padded_map_idx);
        for (unsigned int i = 0; i < substream_padded_map_idx_size; i++) {
            logger_->debug("substream_padded_map_idx={}", dbg_h_padded_map_idx[i]);
        }
#endif
        timer.reset();
#endif

        if (substream_padded_map_idx_size == 0) {
#ifdef DEBUG_OUTPUT
            logger_->debug("no map to be padded");
#endif
            continue;
        }

        // allocate keystore
        stream_data->keystore->len = substream_padded_map_idx_size ;
        hipHostAlloc(&(stream_data->keystore->buf), stream_data->keystore->len *
                sizeof(cudautils::Keypoint), hipHostMallocPortable);

        // only calculate location and save keypoints
        if (sift_params_.skipDescriptor) {
#ifdef DEBUG_OUTPUT
            logger_->debug("Skip calculatation of descriptors");
#endif
            // transfer index map to host for referencing correct index
            unsigned int *h_padded_map_idx;
            cudaSafeCall(hipHostAlloc((void **) &h_padded_map_idx, 
                        substream_padded_map_idx_size * sizeof(unsigned int),
                        hipHostMallocPortable));

            cudaSafeCall(hipMemcpyAsync(
                    h_padded_map_idx,
                    substream_padded_map_idx,
                    substream_padded_map_idx_size * sizeof(unsigned int),
                    hipMemcpyDeviceToHost, stream_data->stream));

            // make sure all async memcpys (above) are finished before access
            cudaSafeCall(hipStreamSynchronize(stream_data->stream));

            // save data for all streams to global Sift object store
            for (int i = 0; i < substream_padded_map_idx_size; i++) {

                Keypoint temp;

                unsigned int padding_x;
                unsigned int padding_y;
                unsigned int padding_z;
                ind2sub(x_sub_stride_, y_sub_stride_, h_padded_map_idx[i], padding_x, padding_y, padding_z);
                // correct for dw_ padding, matlab is 1-indexed
                temp.x = x_sub_start + padding_x - dw_ + 1;
                temp.y = y_sub_start + padding_y - dw_ + 1;
                temp.z = padding_z - dw_ + 1;

                stream_data->keystore->buf[i] = temp;
            }
            cudaSafeCall(hipFree(substream_padded_map_idx));
            continue; // do this for every substream forloop
        }

        /*
        Create an array to hold each descriptor ivec vector on VRAM 
        essentially a matrix of substream_padded_map_idx_size by descriptor length
        */
        double *descriptors, *yy_scratch, *ori_scratch;
        uint16_t * idx_scratch, *ori_idx_scratch;
        long desc_mem_size = sift_params_.descriptor_len * 
            substream_padded_map_idx_size * sizeof(double);
        cudaSafeCall(hipMalloc(&descriptors, desc_mem_size));
        // default nFaces 80; 640 bytes per keypoint yy
        cudaSafeCall(hipMalloc(&yy_scratch, sift_params_.nFaces *
                    substream_padded_map_idx_size * sizeof(double)));
        //  160 bytes per keypoint idx
        cudaSafeCall(hipMalloc(&idx_scratch, sift_params_.nFaces *
                    substream_padded_map_idx_size * sizeof(uint16_t)));
        if (sift_params_.TwoPeak_Flag) {
            // default nFaces=80
            cudaSafeCall(hipMalloc(&ori_idx_scratch, sift_params_.nFaces *
                        substream_padded_map_idx_size * sizeof(uint16_t)));
            cudaSafeCall(hipMalloc(&ori_scratch, sift_params_.nFaces *
                        substream_padded_map_idx_size * sizeof(double)));
        }

        //FIXME num_threads should not be hardcoded
        // One keypoint per thread, one thread per block
        unsigned int num_threads = 1;
        // round up by number of threads per block, to calc num of blocks
        unsigned int num_blocks = get_num_blocks(substream_padded_map_idx_size, num_threads);

#ifdef DEBUG_OUTPUT
        /*cudaSafeCall(hipStreamSynchronize(stream_data->stream));*/
        logger_->debug("num_blocks={}", num_blocks);
        logger_->debug("num_threads={}", num_threads);
#endif

        if (num_blocks * num_threads < substream_padded_map_idx_size) {
            logger_->info("Error occured in numblocks and num_threads estimation... returning from stream"); 
            return;
        }

#ifdef DEBUG_OUTPUT
        logger_->debug("create_descriptor");
#endif

        // sift_params.fv_centers must be placed on device since array passed to cuda kernel
        double* device_centers;
        // default fv_centers_len 80 * 3 (3D) = 240;
        cudaSafeCall(hipMalloc((void **) &device_centers,
                    sizeof(double) * sift_params_.fv_centers_len));
        cudaSafeCall(hipMemcpy((void *) device_centers, (const void *) fv_centers_,
                (size_t) sizeof(double) * sift_params_.fv_centers_len,
                hipMemcpyHostToDevice));
        
#ifdef DEBUG_OUTPUT_MATRIX

        printf("Print image\n");
        hipStreamSynchronize(stream_data->stream);
        int sub_volume_size = x_sub_stride_ * y_sub_stride_ * z_stride_;
        double* dbg_h_image = (double*) malloc(sizeof(double) * sub_volume_size);
        cudaSafeCall(hipMemcpy((void **) dbg_h_image, subdom_data->padded_image,
                sizeof(double) * sub_volume_size,
                hipMemcpyDeviceToHost));
        // print
        for (int i=0; i < sub_volume_size; i++) {
            if (dbg_h_image[i] != 0.0) {
                printf("host image[%d]: %f\n", i, dbg_h_image[i]);
            }
        }

#endif

        create_descriptor<<<num_blocks, num_threads, 0, stream_data->stream>>>(
                x_sub_stride_, y_sub_stride_, x_sub_start, y_sub_start, 
                dw_, // pad width
                substream_padded_map_idx_size, // total number of keypoints to process
                substream_padded_map_idx, //substream map, filtered linear idx into per GPU padded_map and padded_image
                subdom_data->padded_map,//global map split per GPU
                subdom_data->padded_image,//image split per GPU
                sift_params_, 
                device_centers,
                descriptors,
                idx_scratch,
                yy_scratch,
                ori_idx_scratch,
                ori_scratch); 
        cudaCheckError();


#ifdef DEBUG_OUTPUT
        logger_->info("create descriptors elapsed: {}", timer.get_laptime());

        timer.reset();
#endif

        // transfer vector descriptors via host pinned memory for faster async cpy
        double *h_descriptors;
        cudaSafeCall(hipHostAlloc((void **) &h_descriptors, desc_mem_size, hipHostMallocPortable));
        
        cudaSafeCall(hipMemcpyAsync(
                h_descriptors,
                descriptors,
                desc_mem_size,
                hipMemcpyDeviceToHost, stream_data->stream));

        // transfer index map to host for referencing correct index
        unsigned int *h_padded_map_idx;
        cudaSafeCall(hipHostAlloc((void **) &h_padded_map_idx, 
                    substream_padded_map_idx_size * sizeof(unsigned int),
                    hipHostMallocPortable));

        cudaSafeCall(hipMemcpyAsync(
                h_padded_map_idx,
                substream_padded_map_idx,
                substream_padded_map_idx_size * sizeof(unsigned int),
                hipMemcpyDeviceToHost, stream_data->stream));

#ifdef DEBUG_OUTPUT_MATRIX
        /*for (int i=0; i < substream_padded_map_idx_size; i++) {*/
            /*printf("h_padded_map_idx:%u\n", h_padded_map_idx[i]);*/
            /*if (i % sift_params_.descriptor_len == 0) {*/
                /*printf("\n\nDescriptor:%d\n", (int) i / sift_params_.descriptor_len);*/
            /*}*/
            /*printf("%d: %d\n", i, h_descriptors[i]);*/
        /*}*/
#endif

        // make sure all async memcpys (above) are finished before access
        cudaSafeCall(hipStreamSynchronize(stream_data->stream));

        // save data for all streams to global Sift object store
        int skip_counter = 0;
        for (int i = 0; i < substream_padded_map_idx_size; i++) {
            if (std::isnan(h_padded_map_idx[i])) {
                skip_counter++;
                continue;
            } 

            Keypoint temp;

            unsigned int padding_x;
            unsigned int padding_y;
            unsigned int padding_z;
            ind2sub(x_sub_stride_, y_sub_stride_, h_padded_map_idx[i], padding_x, padding_y, padding_z);
            // correct for dw_ padding, matlab is 1-indexed
            temp.x = x_sub_start + padding_x - dw_ + 1;
            temp.y = y_sub_start + padding_y - dw_ + 1;
            temp.z = padding_z - dw_ + 1;

            temp.ivec = (double*) malloc(sift_params_.descriptor_len * sizeof(double));
            // FIXME is this faster than individual device to host transfers
            memcpy(temp.ivec, &(h_descriptors[i * sift_params_.descriptor_len]), 
                    sift_params_.descriptor_len * sizeof(double));
            temp.xyScale = sift_params_.xyScale;
            temp.tScale = sift_params_.tScale;

#ifdef DEBUG_OUTPUT_MATRIX
            /*logger_->info("XXX    desc_len={}, x_sub_start={}, y_sub_start={}, idx={}, temp.x={}, temp.y={}, temp.z={}",*/
                    /*sift_params_.descriptor_len, x_sub_start, y_sub_start, h_padded_map_idx[i], temp.x,*/
                    /*temp.y, temp.z);*/
            for (int desc_idx=0; desc_idx < sift_params_.descriptor_len; desc_idx++) {
                logger_->info("ivec[{}]={}", desc_idx, temp.ivec[desc_idx]);
            }
#endif

            // buffer the size of the whole image
            stream_data->keystore->buf[i - skip_counter] = temp;
        }

        // remove rejected keypoints
        auto new_end = thrust::remove_if(thrust::device,
                stream_data->keystore->buf, 
                stream_data->keystore->buf + stream_data->keystore->len,
                h_padded_map_idx, isnan_test());
        // update the len for transfer
        stream_data->keystore->len = substream_padded_map_idx_size -
            skip_counter;

#ifdef DEBUG_OUTPUT
        cudaSafeCall(hipStreamSynchronize(stream_data->stream));
        logger_->info("stream_data->keystore->len={}, new_end - stream_data->keystore->buf={}",
                stream_data->keystore->len, new_end - stream_data->keystore->buf);
#endif
        assert(stream_data->keystore->len == (new_end - stream_data->keystore->buf));

        cudaSafeCall(hipFree(substream_padded_map_idx));
        cudaSafeCall(hipFree(descriptors));
        cudaSafeCall(hipFree(device_centers));
        cudaSafeCall(hipFree(idx_scratch));
        cudaSafeCall(hipFree(yy_scratch));
        if (sift_params_.TwoPeak_Flag) {
            cudaSafeCall(hipFree(ori_idx_scratch));
            cudaSafeCall(hipFree(ori_scratch));
        }

        cudaSafeCall(hipHostFree(h_descriptors));
        cudaSafeCall(hipHostFree(h_padded_map_idx));

#ifdef DEBUG_OUTPUT
        logger_->info("transfer d2h and copy descriptor ivec values {}", timer.get_laptime());

#endif
    }
}


} // namespace cudautils

