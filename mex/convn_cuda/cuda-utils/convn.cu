#include <hipDNN.h>
#include <iostream>
#include <cstdlib>

#include "convn.h"

namespace cudautils {

#define checkCUDNN(expression)                                     \
    {                                                              \
        hipdnnStatus_t status = (expression);                       \
        if (status != HIPDNN_STATUS_SUCCESS) {                      \
            std::cerr << "Error on line " << __LINE__ << ": "      \
                      << hipdnnGetErrorString(status) << std::endl; \
            std::exit(EXIT_FAILURE);                               \
        }                                                          \
    }

float* convn(float *image, const int channels, const int height, const int width, float *kernel, const int kernel_channels, const int kernel_height, const int kernel_width)  {
    // process args
    // input, kernel, and output descriptors must all have the same dimensions : 3
    int batch_size = 1;
    int out_channels = 3; //feature maps
    int in_channels = 3;
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    // define input tensor
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                /*format=*/HIPDNN_TENSOR_NCHW,
                /*dataType=*/HIPDNN_DATA_FLOAT,
                /*batch_size=*/batch_size,
                /*channels=*/channels,
                /*image_height=*/height,
                /*image_width=*/width));

    /*Real code*/
    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                /*dataType=*/HIPDNN_DATA_FLOAT,
                /*format=*/HIPDNN_TENSOR_NCHW, // for defining weights
                /*out_channels=*/out_channels,
                /*in_channels=*/in_channels,
                /*kernel_height=*/kernel_height,
                /*kernel_width=*/kernel_width));

    // describe the convolution
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    // define the zero padding size for each dimension
    const int padA[] = {1, 1, 1};
    // for each dim, define num elements to stride for eacr point
    const int filterStrideA[] = {1, 1, 1};
    const int dilationA[] = {1, 1, 1}; // dilation factor
    const int dimensions = 3; // number of dimensions of the convolution / input matrix
    checkCUDNN(hipdnnSetConvolutionNdDescriptor(convolution_descriptor,
            /*array_length=*/dimensions,
            /*padA=*/padA,
            /*filterStrideA=*/filterStrideA,
            /*dilationA=*/dilationA,
            // /*mode=*/HIPDNN_CONVOLUTION,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));

    /*// get output dimensions of convolutions for allocating correct space*/
    /*// holds size of output tensor*/
    /*// output dims of this function must be strictly respected for `hipdnnConvolutionForward()`*/
    /*int tensorOutputDimA[] = {0, 0, 0};*/
    /*checkCUDNN(cudnnGetConvolutionNdForwardOutputDim(convolution_descriptor,*/
                /*input_descriptor,*/
                /*kernel_descriptor,*/
                /*[>nbDims=<]dimensions,*/
                /*tensorOutputDimA));*/

    /*std::cout << "Output dimensions: " << tensorOutputDimA[0] << ", " << tensorOutputDimA[1] << ", " << tensorOutputDimA[2] << std::endl;*/

    std::cout << "Output dimensions: " << channels << ", " << height << ", " << width << std::endl;

    //  output tensor
    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                /*format=*/HIPDNN_TENSOR_NCHW,
                /*dataType=*/HIPDNN_DATA_FLOAT,
                /*batch_size=*/batch_size,
                /*channels=*/channels,
                /*image_height=*/height,
                /*image_width=*/width));


    /*std::cout << "Starting convolution algorithm" << std::endl;*/
    /*// choose convolution algorithm*/
    /*hipdnnConvolutionFwdAlgo_t convolution_algorithm;*/
    /*size_t memoryLimitInBytes = 0;*/
    /*// allow unlimited memory*/
    /*checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,*/
            /*input_descriptor,*/
            /*kernel_descriptor,*/
            /*convolution_descriptor,*/
            /*output_descriptor,*/
            /*HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,*/
            /*0, */
            /*&convolution_algorithm)); // save chosen algo*/
    hipdnnConvolutionFwdAlgo_t convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;

    // print convolution_algorithm to stdout
    std::cout << "Convolution algorithm: " << convolution_algorithm << std::endl;

    // choose workspace byte size
   std::cout << "Getting workspace size" << std::endl;
   size_t workspace_bytes{0};
   checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
               input_descriptor,
               kernel_descriptor,
               convolution_descriptor,
               output_descriptor,
               convolution_algorithm,
               &workspace_bytes));
   std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;

   // allocation of mem buffers
   void* d_workspace{nullptr};
   hipMalloc(&d_workspace, workspace_bytes);

   /*int image_bytes = batch_size * tensorOutputDimA[0] * tensorOutputDimA[1] * tensorOutputDimA[2] * sizeof(float);*/
   int image_bytes = batch_size * channels * height * width * sizeof(float);

   float* d_input{nullptr};
   hipMalloc(&d_input, image_bytes);
   // copy to GPU device
   hipMemcpy(d_input, image, image_bytes, hipMemcpyHostToDevice);

   float* d_output{nullptr};
   hipMalloc(&d_output, image_bytes);
   // guarantee all vals 0
   hipMemset(d_output, 0, image_bytes);

   float* d_kernel{nullptr};
   hipMalloc(&d_kernel, sizeof(kernel));
   // copy to device
   hipMemcpy(d_kernel, kernel, sizeof(kernel), hipMemcpyHostToDevice);

   // Convolution
   std::cout << "Compute convolution" << std::endl;
   const float alpha = 1.0f;
   const float beta = 0.0f;
   checkCUDNN(hipdnnConvolutionForward(cudnn,
               &alpha, 
               input_descriptor, 
               d_input,
               kernel_descriptor,
               d_kernel,
               convolution_descriptor,
               convolution_algorithm,
               d_workspace,
               workspace_bytes,
               &beta,
               output_descriptor,
               d_output));

   // Copy data back to host
   float* h_output = new float[image_bytes];
   hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

   // free
   // delete[] h_output
   hipFree(d_kernel);
   hipFree(d_input);
   hipFree(d_output);
   hipFree(d_workspace);

   hipdnnDestroyTensorDescriptor(input_descriptor);
   hipdnnDestroyTensorDescriptor(output_descriptor);
   hipdnnDestroyFilterDescriptor(kernel_descriptor);
   hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

   hipdnnDestroy(cudnn);
   std::cout << "Success, exiting" << std::endl;

   return h_output;
}

} //namespace
